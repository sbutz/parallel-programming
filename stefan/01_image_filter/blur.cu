#include "hip/hip_runtime.h"
#include "jpeg.h"
#include "util.h"
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void Blur(unsigned char* inputImage, unsigned char* outputImage, int width, int height, int channels, int margin) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = threadIdx.z;

    if (x < width && y < height) {
        int startX = max(x - margin, 0);
        int endX = min(x + margin, width);
        int startY = max(y - margin, 0);
        int endY = min(y + margin, height);

        float v = 0;
        for (int i = startY; i < endY; i++)
        {
            for (int j = startX; j < endX; j++)
            {
                v += inputImage[(i * width + j) * channels + channel];
            }
        }
        float n = (endX - startX) * (endY - startY);
        outputImage[(y * width + x) * channels + channel] = v / n;
    }
}

int main(int argc, char* argv[])
{
    if (argc != 4)
    {
        std::cerr << "Usage: " << argv[0] << " margin input.jpg output.jpg" << std::endl;
        return 1;
    }

    int margin = std::atoi(argv[1]);
    const char* inputFilename = argv[2];
    const char* outputFilename = argv[3];

    auto hInputImage = Jpeg::FromFile(inputFilename);
    auto height = hInputImage.GetHeight();
    auto width = hInputImage.GetWidth();
    auto channels = hInputImage.GetChannels();

    unsigned char *dInputImage, *dOutputImage;
    CUDA_ASSERT(hipMalloc((void**)&dInputImage, width * height * channels * sizeof(float)));
    CUDA_ASSERT(hipMalloc((void**)&dOutputImage, width * height * channels * sizeof(float)));

    float time;
    hipEvent_t start, stop;
    CUDA_ASSERT(hipEventCreate(&start) );
    CUDA_ASSERT(hipEventCreate(&stop) );
    CUDA_ASSERT(hipEventRecord(start, 0) );

    CUDA_ASSERT(hipMemcpy(dInputImage, hInputImage.GetRawData(), width * height * channels, hipMemcpyHostToDevice));

    CUDA_ASSERT(hipEventRecord(stop, 0) );
    CUDA_ASSERT(hipEventSynchronize(stop) );
    CUDA_ASSERT(hipEventElapsedTime(&time, start, stop) );

    std::cout << "Elapsed time: " << time << "ms" << std::endl;


    dim3 blockSize(16, 16, channels);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    Blur<<<gridSize, blockSize>>>(dInputImage, dOutputImage, width, height, channels, margin);
    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    Jpeg hOutputImage{width, height, channels};
    CUDA_ASSERT(hipMemcpy(hOutputImage.GetRawData(), dOutputImage, width * height * channels, hipMemcpyDeviceToHost));
    hOutputImage.Save(outputFilename);

    CUDA_ASSERT(hipFree(dInputImage));
    CUDA_ASSERT(hipFree(dOutputImage));

    return 0;
}
