#include "hip/hip_runtime.h"
#include "bfs.h"
#include "device_vector.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

DeviceGraph::DeviceGraph(IdxType nVertices, IdxType lenDestinations, IdxType * d_startIndices, IdxType * d_incidenceAry) {
    g.nVertices = nVertices;
    CUDA_CHECK(hipMalloc(&g.d_startIndices, (nVertices + 1) * sizeof(IdxType)))
    CUDA_CHECK(hipMemcpy(g.d_startIndices, d_startIndices, (nVertices + 1) * sizeof(IdxType), hipMemcpyHostToDevice))
    CUDA_CHECK(hipMalloc(&g.d_incidenceAry, lenDestinations * sizeof(IdxType)))
    CUDA_CHECK(hipMemcpy(g.d_incidenceAry, d_incidenceAry, lenDestinations * sizeof(IdxType), hipMemcpyHostToDevice))
}

DeviceGraph::~DeviceGraph() {
    (void)hipFree(g.d_incidenceAry);
    (void)hipFree(g.d_startIndices);
}



static __device__ void appendToFrontier(IdxType * cntFrontier, IdxType * frontier, IdxType vertex) {
    IdxType old = atomicAdd(cntFrontier, 1);
    frontier[old] = vertex;
}

static __global__ void kernel_bfs(
    DNeighborGraph graph,
    unsigned int * d_visited,
    unsigned int visitedTag, // must be != 0
    IdxType * cntFrontier,
    IdxType * frontier,
    IdxType * cntNewFrontier,
    IdxType * newFrontier
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    IdxType stride = 1;

    for (IdxType i = tid * stride; i < tid * stride + 1; ++i) {
        if (i < *cntFrontier) {
            IdxType vertex = frontier[i];
            IdxType incidenceListStart = graph.d_startIndices[vertex];
            IdxType incidenceListEnd = graph.d_startIndices[vertex+1];

            for (IdxType j = incidenceListStart; j < incidenceListEnd; ++j) {
                IdxType destination = graph.d_incidenceAry[j];
                unsigned int destinationVisited = d_visited[destination];
                if(destinationVisited <= 1) {
                    d_visited[destination] = visitedTag;
                    if (destinationVisited == 0) appendToFrontier(cntNewFrontier, newFrontier, destination);
                }
            }

        }
    }
}

ComponentFinder::ComponentFinder(DNeighborGraph const * graph, std::size_t maxFrontierSize) : nVertices(graph->nVertices) {
    // TODO: Should we malloc everything at once?
    CUDA_CHECK(hipMalloc(&this->d_visited, (std::size_t)graph->nVertices * sizeof(IdxType)))
    CUDA_CHECK(hipMemset(this->d_visited, 0, (std::size_t)graph->nVertices * sizeof(IdxType)))

    size_t frontierBufferSize = 2 * (1 + (std::size_t)maxFrontierSize);
    CUDA_CHECK(hipMalloc(&this->d_frontierBuffer, frontierBufferSize * sizeof(IdxType)))
    CUDA_CHECK(hipMemset(this->d_frontierBuffer, 0, frontierBufferSize * sizeof(IdxType)))
    this->frontiers[0] = { this->d_frontierBuffer, this->d_frontierBuffer + 1 };
    this->frontiers[1] = { this->d_frontierBuffer + frontierBufferSize / 2, this->d_frontierBuffer + frontierBufferSize / 2 + 1 };
}

ComponentFinder::~ComponentFinder() {
    (void)hipFree(this->d_frontierBuffer);
    (void)hipFree(this->d_visited);
}

void ComponentFinder::findComponent(
    DNeighborGraph const * graph, IdxType startVertex, IdxType visitedTag,
    void (*callback) (void *), void * callbackData
) {
    constexpr int threadsPerBlock = 128;
    IdxType startValues [2] = { 1, startVertex };

    CUDA_CHECK(hipMemcpy(this->frontiers[this->currentFrontier].d_cntFrontier, &startValues, 2 * sizeof(IdxType), hipMemcpyHostToDevice))
    CUDA_CHECK(hipMemset(&this->d_visited[startVertex], visitedTag, sizeof(IdxType)))
    for (;;) {
        CUDA_CHECK(hipMemset(this->frontiers[!this->currentFrontier].d_cntFrontier, 0, sizeof(IdxType)))
        kernel_bfs <<<
            dim3((graph->nVertices + threadsPerBlock - 1) / threadsPerBlock),
            dim3(threadsPerBlock)
        >>> (
            *graph,
            this->d_visited,
            visitedTag,
            this->frontiers[this->currentFrontier].d_cntFrontier,
            this->frontiers[this->currentFrontier].d_frontier,
            this->frontiers[!this->currentFrontier].d_cntFrontier,
            this->frontiers[!this->currentFrontier].d_frontier
        );
        hipDeviceSynchronize();

        if (callback) (*callback) (callbackData);

        IdxType cntNewFrontier;
        CUDA_CHECK(hipMemcpy(
            &cntNewFrontier, this->frontiers[!this->currentFrontier].d_cntFrontier, sizeof(IdxType),
            hipMemcpyDeviceToHost
        ))
        if (!cntNewFrontier) break;
        this->currentFrontier = !this->currentFrontier;
    }  
}

static __global__ void kernel_markNonCore(
    IdxType * d_visited,
    IdxType * d_d_startIndices,
    IdxType nVertices
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nVertices) {
        if (d_d_startIndices[tid + 1] - d_d_startIndices[tid] == 0) {
            d_visited[tid] = 1;
        }
    }    
}

AllComponentsFinder::AllComponentsFinder(DNeighborGraph const * graph, std::size_t maxFrontierSize)
: cf (graph, maxFrontierSize), nextFreeTag(2), nextStartIndex(0) {
    CUDA_CHECK(hipMalloc(&this->d_resultBuffer, 2 * sizeof(IdxType)))
}


AllComponentsFinder::~AllComponentsFinder() {
    (void)hipFree(this->d_resultBuffer);
}

std::vector<IdxType> AllComponentsFinder::getComponentTagsVector() const {
    auto res = std::vector<IdxType> (this->cf.nVertices);
    CUDA_CHECK(hipMemcpy(res.data(), this->cf.d_visited, this->cf.nVertices * sizeof(IdxType), hipMemcpyDeviceToHost))
    return res;
}

static void markNonCore(AllComponentsFinder * acf, DNeighborGraph const * graph) {
    constexpr int threadsPerBlock = 128;
    kernel_markNonCore <<<
        dim3((graph->nVertices + threadsPerBlock - 1) / threadsPerBlock),
        dim3(threadsPerBlock)    
    >>> (
        acf->cf.d_visited,
        graph->d_startIndices,
        graph->nVertices
    );
    CUDA_CHECK(hipGetLastError())
}

static __global__ void kernel_findUnvisitedNaive(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // TODO: This is bad. We should start at startPos and finish early when vertex has been found.
    if (tid < nVertices) {
        if (!d_visited[tid]) {
            outBuffer[0] = 1; // true
            outBuffer[1] = tid;
        }
    }    
}


template <int FindNextUnvisitedPolicyKey> struct FindNextUnvisitedPolicy;

template <>
struct FindNextUnvisitedPolicy<findNextUnivisitedNaivePolicy> {

    static auto findNextUnvisited(
        IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
    ) {
        constexpr int threadsPerBlock = 128;
        CUDA_CHECK(hipMemset(d_resultBuffer, 0, 2 * sizeof(IdxType)))

        kernel_findUnvisitedNaive <<<
            dim3((nVertices + threadsPerBlock - 1) / threadsPerBlock),
            dim3(threadsPerBlock)
        >>> (d_resultBuffer, d_visited, nVertices, startIdx);
        hipDeviceSynchronize();

        IdxType localBuffer [2];
        CUDA_CHECK(hipMemcpy(localBuffer, d_resultBuffer, 2 * sizeof(IdxType), hipMemcpyDeviceToHost))

        struct Result {
            bool wasFound;
            IdxType idx;
        };
        return Result{!!localBuffer[0], localBuffer[1]};
    }
};

static __global__ void kernel_findUnvisitedSuccessive(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    constexpr int logWrp = 5;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;
    unsigned int tid = threadIdx.x;
    unsigned int idx = (startPos & ~(wrp - 1)) + tid;

    IdxType contribution;
    for (;;) {
        contribution = idx < startPos || idx >= nVertices || !!d_visited[idx] ?
            maxIdxType : idx;

        #pragma unroll
        for (int delta = 1; delta < wrp; delta <<= 1) {
            auto other = __shfl_down_sync(0xffffffff, contribution, delta);
            if (other < contribution) contribution = other;
        }

        contribution = __shfl_sync(0xffffffff, contribution, 0);

        if ((idx >> logWrp) == (nVertices >> logWrp) || contribution != maxIdxType) break;

        idx += wrp;
    };

    if (tid == 0) *outBuffer = contribution;
}

template <>
struct FindNextUnvisitedPolicy<findNextUnivisitedSuccessivePolicy> {

    static auto findNextUnvisited(
        IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
    ) {
        constexpr int threadsPerBlock = 32;
        constexpr int blocks = 1;
        constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

        IdxType localBuffer;
        kernel_findUnvisitedSuccessive <<<
            dim3(blocks), dim3(threadsPerBlock)
        >>> (d_resultBuffer, d_visited, nVertices, startIdx);
        hipDeviceSynchronize();

        CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

        struct Result {
            bool wasFound;
            IdxType idx;
        };
        return Result{localBuffer != maxIdxType, localBuffer};
    }
};



static __global__ void kernel_findUnvisitedSuccessiveSimplified(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    IdxType result = (IdxType)-1;
    for (IdxType strideIdx = startPos / wrp; strideIdx <= ((nVertices - 1) / wrp); ++strideIdx) {
        IdxType idx = strideIdx * wrp + threadIdx.x;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < nVertices && !d_visited[idx]);
        if (unvisitedMask != 0) {
            result = strideIdx * wrp + __ffs(unvisitedMask) - 1;
            break;
        }
    }
    if (threadIdx.x == 0) *outBuffer = result;
}

template <>
struct FindNextUnvisitedPolicy<findNextUnivisitedSuccessiveSimplifiedPolicy> {

    static auto findNextUnvisited(
        IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
    ) {
        struct Result {
            bool wasFound;
            IdxType idx;
        };

        constexpr int threadsPerBlock = 32;
        constexpr int blocks = 1;
        constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

        if (startIdx >= nVertices) return Result{false, 0};

        IdxType localBuffer;
        kernel_findUnvisitedSuccessiveSimplified <<<
            dim3(blocks), dim3(threadsPerBlock)
        >>> (d_resultBuffer, d_visited, nVertices, startIdx);
        hipDeviceSynchronize();

        CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

        return Result{localBuffer != maxIdxType, localBuffer};
    }
};

static __global__ void kernel_findUnvisitedSuccessiveMultWarp(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    constexpr unsigned int stride = 2 * wrp;
    constexpr unsigned int strideStartMask = ~(stride - 1);
    constexpr int warpsPerBlock = stride / wrp;

    __shared__ unsigned int contributions[warpsPerBlock];

    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    //unsigned int stride = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int wid = threadIdx.x / wrp;
    unsigned int lane = threadIdx.x % wrp;

    IdxType strideStartIdx = (startPos & ~(wrp - 1));

    IdxType contribution;
    for (;;) {
        // ! TODO: this may overflow
        IdxType idx = strideStartIdx + tid;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < nVertices && !d_visited[idx]);

        if (lane == 0) contributions[wid] = unvisitedMask ? strideStartIdx + wrp * wid + __ffs(unvisitedMask) - 1 : maxIdxType;

        __syncthreads();

        if (wid == 0) {
            contribution = tid < warpsPerBlock ? contributions[tid] : maxIdxType;

            #pragma unroll
            for (int delta = 1; delta < warpsPerBlock; delta <<= 1) {
                auto other = __shfl_down_sync(0xffffffff, contribution, delta);
                if (other < contribution) contribution = other;
            }

            if (tid == 0) contributions[0] = contribution;
        }

        __syncthreads();

        contribution = contributions[0];

        if (strideStartIdx >= (nVertices & strideStartMask) || contribution != maxIdxType) break;

        strideStartIdx += stride;
    };

    if (tid == 0) *outBuffer = contribution;
}



template <>
struct FindNextUnvisitedPolicy<findNextUnivisitedSuccessiveMultWarpPolicy> {

    static auto findNextUnvisited(
        IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
    ) {
        constexpr int threadsPerBlock = 2 * 32;
        constexpr int blocks = 1;
        constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

        IdxType localBuffer;
        kernel_findUnvisitedSuccessiveMultWarp <<<
            dim3(blocks), dim3(threadsPerBlock)
        >>> (d_resultBuffer, d_visited, nVertices, startIdx);
        CUDA_CHECK(hipGetLastError())
        hipDeviceSynchronize();

        CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

        struct Result {
            bool wasFound;
            IdxType idx;
        };
        return Result{localBuffer != maxIdxType, localBuffer};
    }
};



template <int FindNextUnvisitedPolicyKey>
void doFindAllComponents(
    FindComponentsProfile * profile,
    AllComponentsFinder * acf, DNeighborGraph const * graph,
    void (*callback) (void *), void * callbackData
) {
    profile->timeMarkNonCore = runAndMeasureCuda(markNonCore, acf, graph);
    profile->timeFindComponents = runAndMeasureCuda([&]{
        IdxType nIterations = 0;
        IdxType startIdx = 1;
        for (;;) {
            auto nextUnvisited = FindNextUnvisitedPolicy<FindNextUnvisitedPolicyKey>::findNextUnvisited(
                acf->d_resultBuffer, acf->cf.d_visited, graph->nVertices, startIdx
            );
            if (!nextUnvisited.wasFound) break;
            acf->cf.findComponent(graph, nextUnvisited.idx, acf->nextFreeTag, callback, callbackData);
            startIdx = nextUnvisited.idx + 1;
            ++acf->nextFreeTag;
            ++nIterations;
        }
    });
}

static void forceInstantiation() __attribute__ ((unused));
static void forceInstantiation() {
    doFindAllComponents<findNextUnivisitedNaivePolicy> (nullptr, nullptr, nullptr, nullptr, nullptr);
    doFindAllComponents<findNextUnivisitedSuccessivePolicy> (nullptr, nullptr, nullptr, nullptr, nullptr);
    doFindAllComponents<findNextUnivisitedSuccessiveMultWarpPolicy> (nullptr, nullptr, nullptr, nullptr, nullptr);
    doFindAllComponents<findNextUnivisitedSuccessiveSimplifiedPolicy> (nullptr, nullptr, nullptr, nullptr, nullptr);
}
