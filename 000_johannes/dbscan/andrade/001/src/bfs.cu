#include "hip/hip_runtime.h"
#include "bfs.h"
#include "device_vector.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

template <class T>
void printArray(T * ary, size_t n) {
    if (n == 0) { std::cout << "[ ]\n"; return; }
    std::cout << "[ " << ary[0];
    for (size_t i = 1; i < n; ++i) std::cout << ", " << ary[i];
    std::cout << " ]\n";
}


DeviceGraph::DeviceGraph(IdxType nVertices, IdxType lenDestinations, IdxType * d_startIndices, IdxType * d_incidenceAry) {
    g.nVertices = nVertices;
    CUDA_CHECK(hipMalloc(&g.d_startIndices, (nVertices + 1) * sizeof(IdxType)))
    CUDA_CHECK(hipMemcpy(g.d_startIndices, d_startIndices, (nVertices + 1) * sizeof(IdxType), hipMemcpyHostToDevice))
    CUDA_CHECK(hipMalloc(&g.d_incidenceAry, lenDestinations * sizeof(IdxType)))
    CUDA_CHECK(hipMemcpy(g.d_incidenceAry, d_incidenceAry, lenDestinations * sizeof(IdxType), hipMemcpyHostToDevice))
}

DeviceGraph::~DeviceGraph() {
    (void)hipFree(g.d_incidenceAry);
    (void)hipFree(g.d_startIndices);
}



static __device__ void appendToFrontier(IdxType * cntFrontier, IdxType * frontier, IdxType vertex) {
    IdxType old = atomicAdd(cntFrontier, 1);
    frontier[old] = vertex;
}

static __global__ void kernel_bfs(
    DNeighborGraph graph,
    unsigned int * d_visited,
    unsigned int visitedTag, // must be != 0
    IdxType * cntFrontier,
    IdxType * frontier,
    IdxType * cntNewFrontier,
    IdxType * newFrontier
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    IdxType stride = 1;

    for (IdxType i = tid * stride; i < tid * stride + 1; ++i) {
        if (i < *cntFrontier) {
            IdxType vertex = frontier[i];
            IdxType incidenceListStart = graph.d_startIndices[vertex];
            IdxType incidenceListEnd = graph.d_startIndices[vertex+1];

            for (IdxType j = incidenceListStart; j < incidenceListEnd; ++j) {
                IdxType destination = graph.d_incidenceAry[j];
                unsigned int destinationVisited = d_visited[destination];
                if(destinationVisited <= 1) {
                    d_visited[destination] = visitedTag;
                    if (destinationVisited == 0) appendToFrontier(cntNewFrontier, newFrontier, destination);
                }
            }

        }
    }
}

ComponentFinder::ComponentFinder(DNeighborGraph const * graph, std::size_t maxFrontierSize) : nVertices(graph->nVertices) {
    // TODO: Should we malloc everything at once?
    CUDA_CHECK(hipMalloc(&this->d_visited, (std::size_t)graph->nVertices * sizeof(IdxType)))
    CUDA_CHECK(hipMemset(this->d_visited, 0, (std::size_t)graph->nVertices * sizeof(IdxType)))

    size_t frontierBufferSize = 2 * (1 + (std::size_t)maxFrontierSize);
    CUDA_CHECK(hipMalloc(&this->d_frontierBuffer, frontierBufferSize * sizeof(IdxType)))
    CUDA_CHECK(hipMemset(this->d_frontierBuffer, 0, frontierBufferSize * sizeof(IdxType)))
    this->frontiers[0] = { this->d_frontierBuffer, this->d_frontierBuffer + 1 };
    this->frontiers[1] = { this->d_frontierBuffer + frontierBufferSize / 2, this->d_frontierBuffer + frontierBufferSize / 2 + 1 };
}

ComponentFinder::~ComponentFinder() {
    (void)hipFree(this->d_frontierBuffer);
    (void)hipFree(this->d_visited);
}

void ComponentFinder::findComponent(
    DNeighborGraph const * graph, IdxType startVertex, IdxType visitedTag,
    void (*callback) (void *), void * callbackData
) {
    constexpr int threadsPerBlock = 128;
    IdxType startValues [2] = { 1, startVertex };

    CUDA_CHECK(hipMemcpy(this->frontiers[this->currentFrontier].d_cntFrontier, &startValues, 2 * sizeof(IdxType), hipMemcpyHostToDevice))
    CUDA_CHECK(hipMemset(&this->d_visited[startVertex], visitedTag, sizeof(IdxType)))
    for (;;) {
        CUDA_CHECK(hipMemset(this->frontiers[!this->currentFrontier].d_cntFrontier, 0, sizeof(IdxType)))
        kernel_bfs <<<
            dim3((graph->nVertices + threadsPerBlock - 1) / threadsPerBlock),
            dim3(threadsPerBlock)
        >>> (
            *graph,
            this->d_visited,
            visitedTag,
            this->frontiers[this->currentFrontier].d_cntFrontier,
            this->frontiers[this->currentFrontier].d_frontier,
            this->frontiers[!this->currentFrontier].d_cntFrontier,
            this->frontiers[!this->currentFrontier].d_frontier
        );
        hipDeviceSynchronize();

        if (callback) (*callback) (callbackData);

        IdxType cntNewFrontier;
        CUDA_CHECK(hipMemcpy(
            &cntNewFrontier, this->frontiers[!this->currentFrontier].d_cntFrontier, sizeof(IdxType),
            hipMemcpyDeviceToHost
        ))
        if (!cntNewFrontier) break;
        this->currentFrontier = !this->currentFrontier;
    }  
}

static __global__ void kernel_markNonCore(
    IdxType * d_visited,
    IdxType * d_d_startIndices,
    IdxType nVertices
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nVertices) {
        if (d_d_startIndices[tid + 1] - d_d_startIndices[tid] == 0) {
            d_visited[tid] = 1;
        }
    }    
}

static __global__ void kernel_findUnvisited(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // TODO: This is bad. We should start at startPos and finish early when vertex has been found.
    if (tid < nVertices) {
        if (!d_visited[tid]) {
            outBuffer[0] = 1; // true
            outBuffer[1] = tid;
        }
    }    
}

AllComponentsFinder::AllComponentsFinder(DNeighborGraph const * graph, std::size_t maxFrontierSize)
: cf (graph, maxFrontierSize), nextFreeTag(2), nextStartIndex(0) {
    CUDA_CHECK(hipMalloc(&this->d_resultBuffer, 2 * sizeof(IdxType)))
}


AllComponentsFinder::~AllComponentsFinder() {
    (void)hipFree(this->d_resultBuffer);
}

std::vector<IdxType> AllComponentsFinder::getComponentTagsVector() const {
    auto res = std::vector<IdxType> (this->cf.nVertices);
    CUDA_CHECK(hipMemcpy(res.data(), this->cf.d_visited, this->cf.nVertices * sizeof(IdxType), hipMemcpyDeviceToHost))
    return res;
}

static void markNonCore(AllComponentsFinder * acf, DNeighborGraph const * graph) {
    constexpr int threadsPerBlock = 128;
    kernel_markNonCore <<<
        dim3((graph->nVertices + threadsPerBlock - 1) / threadsPerBlock),
        dim3(threadsPerBlock)    
    >>> (
        acf->cf.d_visited,
        graph->d_startIndices,
        graph->nVertices
    );
    CUDA_CHECK(hipGetLastError())
}

static auto findNextUnvisited(AllComponentsFinder * acf, DNeighborGraph const * graph) {
    constexpr int threadsPerBlock = 128;
    CUDA_CHECK(hipMemset(acf->d_resultBuffer, 0, 2 * sizeof(IdxType)))

    kernel_findUnvisited <<<
        dim3((graph->nVertices + threadsPerBlock - 1) / threadsPerBlock),
        dim3(threadsPerBlock)
    >>> (acf->d_resultBuffer, acf->cf.d_visited, graph->nVertices, 0);
    hipDeviceSynchronize();

    IdxType localBuffer [2];
    CUDA_CHECK(hipMemcpy(localBuffer, acf->d_resultBuffer, 2 * sizeof(IdxType), hipMemcpyDeviceToHost))

    struct Result {
        bool wasFound;
        IdxType idx;
    };
    return Result{!!localBuffer[0], localBuffer[1]};
}

void doFindAllComponents(
    AllComponentsFinder * acf, DNeighborGraph const * graph,
    void (*callback) (void *), void * callbackData
) {
    markNonCore(acf, graph);
    for (;;) {
        auto nextUnvisited = findNextUnvisited(acf, graph);
        if (!nextUnvisited.wasFound) return;
        acf->cf.findComponent(graph, nextUnvisited.idx, acf->nextFreeTag, callback, callbackData);
        ++acf->nextFreeTag;
    }
}

