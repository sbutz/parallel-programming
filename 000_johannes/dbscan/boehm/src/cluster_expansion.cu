#include "hip/hip_runtime.h"
#include "cluster_expansion.h"
#include "types.h"
#include "cuda_helpers.h"

#include <iostream>
#include <hip/hip_runtime.h>

DPoints copyPointsToDevice(float const * x, float const * y, IdxType n) {
  float * d_x, * d_y;
  CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)))
  CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)))
  CUDA_CHECK(hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice))
  CUDA_CHECK(hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice))
  return { n, d_x, d_y };
}

constexpr IdxType maxSeedLength = 1024; // TODO: Could be larger!?
constexpr IdxType nChains = 512; // TODO: ??


struct ThreadData {
  IdxType currentClusterId;
  unsigned int threadGroupIdx;
  IdxType pointBeingProcessedIdx;
  IdxType * clusters;
  unsigned int * pointStates;
  IdxType * seedList;
  IdxType * seedClusterIds;
  IdxType * seedLength;
  IdxType * seedReserved;
  IdxType * neighborBuffer;
  IdxType * neighborCount;
  bool * s_collisions;
};

// collisionMatrix:
//   in theory: n x n, but only entries (i,j) with j >= i used
//   so entry (i,j) is preceded by how many entries?
//   n for row 0
//   n-1 for row 1
//   ...
//   n-i+1 for row (i-1)
//   j-i for row i
//   -> i * (n + n - i + 1)/2 + j - i = j + i * (2n - i + 1 - 2) / 2 = j + i * (2 * n - (i + 1)) / 2
// correct?
//   entry (0, n-1) -> n-1 + 0
//   entry (1, 1) -> 1 + 1 * (2n - 2) / 2 -> n
//   entry (2, 2) -> 2 + 2 * (2n - 3) / 2 -> 2n - 1
//         (i, i) -> i * n - i * (i - 1) / 2
//         (i, j) -> i * n - i * (i + 1) / 2 + j
static __device__ void markAsCandidate(
  ThreadData const & td,
  IdxType pointIdx
) {
  unsigned int state = td.pointStates[pointIdx];
  unsigned int potentiallyFreeMask = __ballot_sync(__activemask(), state == stateFree);
  if (state == stateFree) {
    int lane = threadIdx.x % 32;
    int leader = __ffs(potentiallyFreeMask) - 1;
    int nPotentiallyFree = __popc(potentiallyFreeMask);
    IdxType oldReserved;
    if (lane == leader) oldReserved = atomicAdd(td.seedReserved, nPotentiallyFree);
    oldReserved = __shfl_sync(potentiallyFreeMask, oldReserved, leader);
    int potentialThreadPosition = __popc(potentiallyFreeMask & ((1u << (lane + 1)) - 1));
    if (oldReserved < maxSeedLength && potentialThreadPosition <= maxSeedLength - oldReserved) {
      state = atomicCAS(&td.pointStates[pointIdx], stateFree, stateReserved);
      unsigned int actuallyFreeMask = __ballot_sync(__activemask(), state == stateFree);
      int nActuallyFree = __popc(actuallyFreeMask);
      int actualLeader = __ffs(actuallyFreeMask) - 1;
      if (state == stateFree) {
        int h;
        if (lane == actualLeader) h = atomicAdd(td.seedLength, __popc(actuallyFreeMask));
        h = __shfl_sync(actuallyFreeMask, h, actualLeader);
        int offset = __popc(actuallyFreeMask & ((1u << lane) - 1));
        td.seedList[h + offset] = pointIdx; td.seedClusterIds[h + offset] = td.currentClusterId;
      }
      if (lane == leader) atomicSub(td.seedReserved, nPotentiallyFree - nActuallyFree);
    } else {
      if (lane == leader) atomicSub(td.seedReserved, nPotentiallyFree);
    }
  }
  switch (state & stateStateBitsMask) {
    case stateCore: {
      // TODO: handle collision!
    } break;
    case stateNoiseOrBorder:
    case stateFree:
    case stateReserved: {
      td.clusters[pointIdx] = td.currentClusterId;
    } break;
    case stateUnderInspection: {
      td.s_collisions[state & stateThreadGroupIdxMask] = true;
      //printf("inspection\n");
      //(void)atomicCAS(&td.clusters[pointIdx], 0, td.currentClusterId);
    } break;
    default:
      ; // do nothing
  }
  /*
  if (oldState == cUnprocessed) {
  } else {
    if (oldState != cNoise && oldState != cChain + blockIdx.x) {
      if (oldState < pointIdx) {
        // (oldState, pointIdx)
        collisionMatrix[pointIdx + oldState * (2 * nChains - (oldState + 1)) / 2] = true;
      } else {
        collisionMatrix[oldState + pointIdx * (2 * nChains - (pointIdx + 1)) / 2] = true;
      }
    }
  }
  */
}

static __device__ void processObject(
  ThreadData const & td,
  float px, float py,
  float const * xs, float const * ys, IdxType pointIdx,
  IdxType coreThreshold, float rsq
) {
  int lane = threadIdx.x % 32;
  float dx = xs[pointIdx] - px;
  float dy = ys[pointIdx] - py;
  bool isNeighbor = dx * dx + dy * dy <= rsq;
  unsigned int neighborMask = __ballot_sync(__activemask(), isNeighbor);
  if (isNeighbor) {
    int leaderLane = __ffs(neighborMask) - 1;
    int nNeighbors = __popc(neighborMask);
    int oldNeighborCount = 0;
    if (lane == leaderLane) oldNeighborCount = atomicAdd(td.neighborCount, nNeighbors);
    oldNeighborCount = __shfl_sync(neighborMask, oldNeighborCount, leaderLane);
    if (oldNeighborCount < coreThreshold && oldNeighborCount + nNeighbors >= coreThreshold && lane == leaderLane) {
      td.clusters[td.pointBeingProcessedIdx] = td.currentClusterId;
      __threadfence();
      td.pointStates[td.pointBeingProcessedIdx] = stateCore;
    }
    int h = oldNeighborCount + __popc(neighborMask & ((1u << lane) - 1));
    if (h >= coreThreshold) {
      markAsCandidate(td, pointIdx);
    } else {
      td.neighborBuffer[h] = pointIdx;
    }
  }
}

// make VS Code happy
__device__ void __nanosleep(unsigned int);

static __device__ __forceinline__ void lockMutex(unsigned int * d_mutex) {
  unsigned int old;
  int ns = 1;
  do {
    old = atomicCAS(d_mutex, 0, 1);
    
    __nanosleep(ns);
    if (ns < 256) ns *= 2;
  } while (!old);
}

static __device__ __forceinline__ void unlockMutex(unsigned int * d_mutex) {
  *d_mutex = 0;
}



// len(seedLists) must equal maxSeedLength * maxNumberOfThreadGroups
// shared memory required: ( (coreThreshold + 127) / 128 * 128 + 1 ) * sizeof(IdxType)
static __global__ void kernel_clusterExpansion(
  IdxType * cluster, unsigned int * pointState,
  float const * xs, float const * ys, IdxType n,
  IdxType * seedLists, unsigned int * seedClusterIds, IdxType * seedLengths,
  unsigned int * syncCounter, CollisionHandlingData collisionHandlingData,
  IdxType coreThreshold, float rsq
) {
  // shared memory:
  //   neighborBuffer: coreThreshold IdxType values in blocks of 128 bytes
  //   neighborCount:  1 IdxType (aligned to a 128 byte block)
  //   seedReserved:   1 IdxType
  //   s_collisions:   nBlock bools, aligned to a 128 byte block
  //   s_doneWithIdx:  nBlock IdxType values, aligned to a 128 byte block
  extern __shared__ unsigned char sMem [];

  unsigned int stride = blockDim.x;
  unsigned int nBlocks = gridDim.x;

  ThreadData td;
  td.threadGroupIdx = blockIdx.x;

  td.clusters = cluster;
  td.pointStates = pointState;

  td.neighborBuffer = (IdxType *)sMem; // Length: coreThreshold elements
  td.neighborCount = (IdxType *) (sMem + (coreThreshold * sizeof(IdxType) + 127) / 128 * 128);
  td.seedReserved = td.neighborCount + 1;
  td.s_collisions = (bool *) ((char *)td.neighborCount + 128);
  IdxType * s_doneWithIdx = (IdxType *) ((char *)td.s_collisions + (nBlocks + 127) / 128 * 128);
  static_assert(128 % alignof(IdxType) == 0, "");
  static_assert(alignof(IdxType) == alignof(unsigned int) && sizeof(IdxType) == sizeof(unsigned int), "");

  if (threadIdx.x == 0) { *td.neighborCount = 0; }
  for (unsigned int i = threadIdx.x; i < nBlocks; i += stride) td.s_collisions[i] = false;

  td.seedLength = &seedLengths[td.threadGroupIdx];
  td.seedList = &seedLists[maxSeedLength * td.threadGroupIdx];
  td.seedClusterIds = &seedClusterIds[maxSeedLength * td.threadGroupIdx];

  IdxType seedLength = *td.seedLength;

  __syncthreads();

  if (seedLength > 0) {
    --seedLength;

    if (threadIdx.x == 0) { *td.seedLength = seedLength; *td.seedReserved = seedLength; }

    __syncthreads();
    
    td.pointBeingProcessedIdx = td.seedList[seedLength];
    td.currentClusterId = td.seedClusterIds[seedLength];
    if (td.currentClusterId == 0) {
      td.currentClusterId = td.pointBeingProcessedIdx + 1;
    }
    if (threadIdx.x == 0) td.pointStates[td.pointBeingProcessedIdx] = stateUnderInspection | td.threadGroupIdx;

    __syncthreads();

    float x = xs[td.pointBeingProcessedIdx], y = ys[td.pointBeingProcessedIdx];
    {
      IdxType strideIdx = 0;
      for (; strideIdx < (n - 1) / stride; ++strideIdx) {
        processObject(
          td,
          x, y,
          xs, ys, strideIdx * stride + threadIdx.x,
          coreThreshold, rsq
        );
      }
      if (threadIdx.x < n - strideIdx * stride) {
        processObject(
          td,
          x, y,
          xs, ys, strideIdx * stride + threadIdx.x,
          coreThreshold, rsq
        );
      }
    }

    __syncthreads();

    if (*td.neighborCount >= coreThreshold) {
      //if (threadIdx.x == 0) pointState[seedPointIdx] = stateCore;
      for (int i = threadIdx.x; i < coreThreshold; i += stride) {
        markAsCandidate(td, td.neighborBuffer[i]);
      }
    } else {
      if (threadIdx.x == 0) td.pointStates[td.pointBeingProcessedIdx] = stateNoiseOrBorder;
    }
  }

  __syncthreads();

  // copy our collisions to global memory
  for (unsigned int i = threadIdx.x; i < nBlocks; i += stride) collisionHandlingData.d_collisionMatrix[nBlocks * td.threadGroupIdx + i] = td.s_collisions[i];
  if (threadIdx.x == 0) { lockMutex(collisionHandlingData.d_mutex); }

  __threadfence();

  for (unsigned int i = threadIdx.x; i < nBlocks; i += stride) s_doneWithIdx[i] = collisionHandlingData.d_doneWithIdx[i];
  if (threadIdx.x == 0) collisionHandlingData.d_doneWithIdx[td.threadGroupIdx] = td.pointBeingProcessedIdx;

  __threadfence();

  if (threadIdx.x == 0) unlockMutex(collisionHandlingData.d_mutex);

  __threadfence();

  for (unsigned int i = threadIdx.x; i < nBlocks; i += stride) {
    if (i != td.threadGroupIdx) {
      IdxType otherIdx = s_doneWithIdx[i];
      if (otherIdx) {
        bool collision = td.s_collisions[i] || collisionHandlingData.d_collisionMatrix[i * nBlocks + td.threadGroupIdx];
        if (collision) {
          IdxType otherIdx = s_doneWithIdx[i];
          //printf("%u\n", otherIdx);
          if (*td.neighborCount >= coreThreshold) {
            // we are core
            if (td.pointStates[otherIdx] == stateCore) {
              // mark conflict in union-find datastructure
            } else {
              td.clusters[otherIdx] = td.currentClusterId;
            }
          } else {
            // we are noise
            if (td.pointStates[otherIdx] == stateCore) {
              td.clusters[td.pointBeingProcessedIdx] = td.clusters[otherIdx];
            }
          }
        }
      }
    }
  }
}

void allocateDeviceMemory(
  unsigned int ** d_pointStates, IdxType ** d_clusters,
  IdxType ** d_seedLists, IdxType ** d_seedClusterIds, IdxType ** d_seedLengths,
  unsigned int ** d_syncCounter, CollisionHandlingData * collisionHandlingData, IdxType ** d_processedIdxs,
  int nBlocks,
  IdxType n
) {
  CUDA_CHECK(hipMalloc(d_pointStates, n * sizeof(unsigned int)))
  // TODO: Change later
  CUDA_CHECK(hipMemset(*d_pointStates, 0, n * sizeof(unsigned int)))
  CUDA_CHECK(hipMalloc(d_clusters, n * sizeof(IdxType)))
  // TODO: Change later
  CUDA_CHECK(hipMemset(*d_clusters, 0, n * sizeof(IdxType)))

  CUDA_CHECK(hipMalloc(d_seedLists, nBlocks * maxSeedLength * sizeof(IdxType)))
  CUDA_CHECK(hipMalloc(d_seedClusterIds, nBlocks * maxSeedLength * sizeof(IdxType)))
  CUDA_CHECK(hipMalloc(d_seedLengths, nBlocks * sizeof(IdxType)))
  CUDA_CHECK(hipMemset(*d_seedLengths, 0, nBlocks * sizeof(IdxType)))
  CUDA_CHECK(hipMalloc(d_syncCounter, sizeof(unsigned int)))

  auto chdSizes = CollisionHandlingData::calculateSizes(nBlocks);
  unsigned int chdTotalSize = chdSizes.szMutex + chdSizes.szDoneWithIdx + chdSizes.szCollisionMatrix;
  char * d_memCollisionData;
  CUDA_CHECK(hipMalloc(&d_memCollisionData, chdTotalSize))
  collisionHandlingData->d_mutex = (unsigned int *)d_memCollisionData;
  CUDA_CHECK(hipMemset(collisionHandlingData->d_mutex, 0, sizeof(unsigned int)))
  collisionHandlingData->d_doneWithIdx = (IdxType *)(d_memCollisionData + chdSizes.szMutex);
  collisionHandlingData->d_collisionMatrix = (bool *)(d_memCollisionData + chdSizes.szMutex + chdSizes.szDoneWithIdx);
  CUDA_CHECK(hipMalloc(d_processedIdxs, nBlocks * sizeof(IdxType)))
}

static __global__ void kernel_populateSeedLists(
  IdxType * d_seedLists, IdxType * d_seedClusterIds, IdxType * d_seedLengths, IdxType nLists
) {
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int stride = gridDim.x * blockDim.x;
  for (unsigned int i = tid; i < nLists; i += stride) {
    d_seedLengths[i] = 1;
    d_seedLists[i * maxSeedLength] = i; // TODO: Change later!!!
    d_seedClusterIds[i * maxSeedLength] = i + 1; // TODO: Change later!!!
  }
}

static __global__ void kernel_refillSeed(
    IdxType * d_foundAt,
    IdxType * d_seedLists, IdxType * d_seedClusterIds, IdxType * d_seedLengths, int k,
    unsigned int * d_pointState, IdxType * d_clusters, IdxType n,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    IdxType result = (IdxType)-1;
    for (IdxType strideIdx = startPos / wrp; strideIdx <= ((n - 1) / wrp); ++strideIdx) {
        IdxType idx = strideIdx * wrp + threadIdx.x;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < n && !d_pointState[idx]);
        if (unvisitedMask != 0) {
            result = strideIdx * wrp + __ffs(unvisitedMask) - 1;
            break;
        }
    }
    if (threadIdx.x == 0) {
      if (result == (IdxType)-1) {
        *d_foundAt = result;
      } else {
        *d_foundAt = result;
        d_pointState[result] = stateReserved2;
        d_seedLists[k * maxSeedLength] = result;
        d_seedClusterIds[k * maxSeedLength] = d_clusters[result];
        d_seedLengths[k] = 1;
      }
    }
}

void findClusters(
  unsigned int * d_pointStates, IdxType * d_clusters,
  float * xs, float * ys, IdxType n,
  IdxType * d_seedLists, IdxType * d_seedClusterIds, IdxType * d_seedLengths,
  unsigned int * d_syncCounter, CollisionHandlingData collisionHandlingData, IdxType * d_processedIdxs,
  IdxType coreThreshold, float rsq
) {
  constexpr int nBlocks = 6;
  constexpr int nThreadsPerBlock = 512;

  IdxType * d_foundAt;
  CUDA_CHECK(hipMalloc(&d_foundAt, sizeof(IdxType)))

  IdxType seedLengths [nBlocks];
  IdxType startPos = 0;
  for (;;) {
    CUDA_CHECK(hipMemcpy(seedLengths, d_seedLengths, nBlocks * sizeof(IdxType), hipMemcpyDeviceToHost))
    bool stillWork = false;

    for (int k = 0; k < nBlocks; ++k) {
      if (seedLengths[k]) {
        stillWork = true;
      } else if (startPos != (IdxType)-1) {
        IdxType foundAt = (IdxType)-1;
        kernel_refillSeed <<<dim3(1), dim3(32)>>> (d_foundAt, d_seedLists, d_seedClusterIds, d_seedLengths, k, d_pointStates, d_clusters, n, startPos);
        CUDA_CHECK(hipGetLastError())
        CUDA_CHECK(hipMemcpy(&foundAt, d_foundAt, sizeof(IdxType), hipMemcpyDeviceToHost))
        CUDA_CHECK(hipDeviceSynchronize())
        // std::cerr << "Refilled " << k << " " << foundAt << "\n";
        startPos = foundAt + (foundAt != (IdxType)-1);
        stillWork = stillWork || foundAt != (IdxType)-1;
      }
    }

    if (!stillWork) break;

    CUDA_CHECK(hipMemset(d_syncCounter, 0, sizeof(unsigned int)))
    CUDA_CHECK(hipMemset((void *)collisionHandlingData.d_doneWithIdx, 0, nBlocks * sizeof(IdxType)))
    kernel_clusterExpansion <<<dim3(nBlocks), dim3(nThreadsPerBlock), ( (coreThreshold * sizeof(IdxType) + 127) / 128 * 128 + 128 
    + (nBlocks * sizeof(bool) + 127) / 128 * 128 + (nBlocks * sizeof(IdxType) + 127) / 128 * 128) >>> (
      d_clusters, d_pointStates, xs, ys, n, d_seedLists, d_seedClusterIds, d_seedLengths, d_syncCounter, collisionHandlingData, coreThreshold, rsq
    );
    CUDA_CHECK(hipGetLastError())
    CUDA_CHECK(hipDeviceSynchronize())
  }
}

