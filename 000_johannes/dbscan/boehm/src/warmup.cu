#include "hip/hip_runtime.h"
#include "warmup.h"
#include "types.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime.h>

static __global__ void prefixScanStep(
  IdxType * dest, IdxType * src, IdxType n, IdxType delta
) {
  // assert: delta <= n
  // assert: delta is a power of 2

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  IdxType s = 0;
  if (n > stride) {
    for (; s < n - stride; s += stride) {
      auto idx = s + tid;
      IdxType res = src[idx];
      if (idx & delta) res += src[( idx & ~(delta - 1u) ) - 1u];
      dest[idx] = res;
    }
  }
  if (tid < n - s) {
    auto idx = s + tid;
    IdxType res = src[idx];
    if (idx & delta) res += src[( idx & ~(delta - 1u) ) - 1u];
    dest[idx] = res;
  }
}

void prefixScanOnDevice(IdxType ** res, IdxType * dest1, IdxType * dest2, IdxType * src, IdxType n) {
  using std::swap;

  if (n == 0) { *res = dest1; return; }

  constexpr unsigned int nThreadsPerBlock = 256;
  dim3 dimBlock(nThreadsPerBlock, 1, 1);
  dim3 dimGrid((n + nThreadsPerBlock - 1) / nThreadsPerBlock, 1, 1);

  prefixScanStep <<<dimGrid, dimBlock>>> (dest1, src, n, 1);
  CUDA_CHECK(hipGetLastError());
  hipDeviceSynchronize();
  // subtract 1 on both sides of the inequality to avoid overflow issues
  //   if leftmost bit of n is 1
  for (IdxType delta = 2; delta - 1 < n - 1; delta <<= 1) {
    swap(dest1, dest2);
    prefixScanStep <<<dimGrid, dimBlock>>> (dest1, dest2, n, delta);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  *res = dest1;
  return;
}

void warmup() {
  constexpr size_t sz64Mi = (size_t)1 << 26;
  IdxType * tempAry, * temp;
  CUDA_CHECK(hipMalloc(&tempAry, 2 * sz64Mi * sizeof(IdxType) ))
  for (int i = 0; i < 5; ++i) {
    prefixScanOnDevice(
      &temp, tempAry, tempAry + sz64Mi, tempAry + sz64Mi, sz64Mi
    );
  }
  (void)hipFree(tempAry);
}