#include "hip/hip_runtime.h"
#include "device_vector.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// return ceil(a / b) without overflow issues
constexpr __host__ __device__ __forceinline__ std::size_t ceilDiv(std::size_t a, std::size_t b) {
  return (a / b) + ((a / b) * b > 0);
}

// return result of ceil(a / b) < c without overflow issues
constexpr __host__ __device__ __forceinline__ bool ceilDivLt(std::size_t a, std::size_t b, std::size_t c) {
  return (a - 1) / b + 1 < c;
}

// return result of a < ceil(b / c) without overflow issues
constexpr __host__ __device__ __forceinline__ bool ltCeilDiv(std::size_t a, std::size_t b, std::size_t c) {
  return (b != 0) && (b - 1) / c >= a;
}

template <int cWarpSize, int nLanes = cWarpSize>
constexpr __host__ __device__ unsigned int getMask() {
  using MaskType = unsigned int;
  // slightly complicated to avoid "shift count is too large" warning
  return (nLanes == cWarpSize ? MaskType{0} : MaskType{1u << nLanes}) - MaskType{1};
}

// The function will perform the necessary synchronization itself.
template <int cWarpSize, int nLanes = cWarpSize>
__device__ float scanPerWarpSync(float v) {
  using MaskType = unsigned int;

  static_assert(nLanes <= cWarpSize, "");
  static_assert(8 * sizeof(MaskType) == cWarpSize, "");

  MaskType constexpr mask = getMask<cWarpSize, nLanes> ();

  unsigned int const lane = threadIdx.x % cWarpSize;
  for (int w = 1; w < nLanes; w <<= 1) {
    float x = __shfl_up_sync(mask, v, w);
    if (lane >= w) v += x;
  }
  return v;
}

// The function will perform the necessary synchronization itself.
template <int cWarpSize>
__device__ float sumToLastOfWarpSync(float v) {
  using MaskType = unsigned int;

  static_assert(8 * sizeof(MaskType) == cWarpSize, "");

  MaskType constexpr mask = MaskType{0} - MaskType{1};

  for (int w = 1; w < cWarpSize; w <<= 1) v += __shfl_up_sync(mask, v, w);
  return v;
}

// Thread cWarpsPerBlock - 1 returns the block sum, the other
//   threads return an undefined value
// block idx will handle values[step * blockDim.x * idx ... step * (blockDim.x * idx + blockDim.x - 1)]
template <int cWarpSize, int cWarpsPerBlock>
struct BlockwiseScan {
  static constexpr std::size_t cBlockSize = cWarpSize * cWarpsPerBlock;

  std::size_t n;
  std::size_t step;

  constexpr __host__ __device__ BlockwiseScan(std::size_t n, std::size_t step) : n(n), step(step) {}

  constexpr __host__ __device__ bool blockDoesParticipate(unsigned int blockIndex) {
    return (n - 1) / cBlockSize > blockIndex;
  }

  constexpr __host__ __device__ std::size_t getLastScannedIndex(unsigned int blockIndex) {
    if ((n - 1) / cBlockSize <= blockIndex + 1) return n - 1;
    return step * (cBlockSize * blockIndex + cBlockSize - 1);
  }

  __device__ float scan (
    float * sTemp,
    float * values
  ) {
    // assert: warpSize == cWarpSize
    // assert: blockDim.x = cWarpSize * cWarpsPerBlock
    static_assert(cWarpsPerBlock <= cWarpSize, "");

    constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

    unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;
    unsigned int const stride = gridDim.x * blockDim.x;
    unsigned int const wid = threadIdx.x / cWarpSize;
    unsigned int const lane = threadIdx.x % cWarpSize;
    unsigned int const warpInGrid = tid / cWarpSize;

    float returnValue = 0;

    if (ltCeilDiv(warpInGrid, n, cWarpSize)) {
      // We calculate the sum for each warp and store these sums in shared memory.
      {
        auto v = tid < n ? values[tid * step] : 0;

        v = sumToLastOfWarpSync<cWarpSize>(v);
        if (lane == cWarpSize - 1) sTemp[wid] = v;
      }

      __syncthreads();

      // Now we have the sums per warp in shared memory.
      // We perform a scan over the shared memory, in order to get
      //   accumulated warpwise sums in shared memory.
      // The scan is performed by the first warp.
      if (threadIdx.x < cWarpsPerBlock) {
        static_assert(cWarpsPerBlock <= cWarpSize,
          "This code is only correct if the per-warp sums within a block "
          "can be scanned by a single warp."
        );

        float v = sTemp[threadIdx.x];
        
        v = scanPerWarpSync<cWarpSize, cWarpsPerBlock> (v);

        sTemp[threadIdx.x] = v;

        returnValue = v;
      }

      return returnValue;
    }
  }
};

static_assert(BlockwiseScan<32, 2>{700, 1}.getLastScannedIndex(0) == 63, "");
static_assert(BlockwiseScan<32, 2>{700, 1}.getLastScannedIndex(1) == 127, "");
static_assert(BlockwiseScan<32, 2>{700, 4}.getLastScannedIndex(1) == 508, "");
static_assert(BlockwiseScan<32, 4>{700, 1}.getLastScannedIndex(0) == 127, "");
static_assert(BlockwiseScan<32, 1>{700, 32}.blockDoesParticipate(0), "");

template <int cWarpSize, int cWarpsPerBlock>
__device__ void scanSingleStrideFillinStep(
  float * dest, std::size_t n, 
  std::size_t step,
  float * sTemp,
  float blockBaseValue,
  float * values
) {
  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock
  static_assert(cWarpsPerBlock <= cWarpSize, "");

  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

  unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;
  unsigned int const stride = gridDim.x * blockDim.x;
  unsigned int const wid = threadIdx.x / cWarpSize;
  unsigned int const lane = threadIdx.x % cWarpSize;
  unsigned int const warpInGrid = tid / cWarpSize;

  using MaskType = unsigned int;
  static_assert(8 * sizeof(MaskType) == cWarpSize, "");

  if (ltCeilDiv(warpInGrid, n, cWarpSize)) {
    auto warpBaseValue = threadIdx.x >= cWarpSize ? sTemp[threadIdx.x / cWarpSize - 1] : 0;
    auto v = tid < n ? values[tid * step] : 0;

    v = scanPerWarpSync<cWarpSize>(v);

    // TODO: Synchronization probably not correct. Reading and writing to dest should be done by the same block.
    if (tid < n && threadIdx.x != cBlockSize - 1)
      dest[tid * step] = blockBaseValue + warpBaseValue + v;
  }
}


template <int cWarpSize, int cWarpsPerBlock>
__forceinline__ __device__ void scanSingleStrideSteps(
  float * dest, std::size_t n, std::size_t nWriteable,
  bool firstStride,
  float * sTemp,
  float * values
) {
  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;
  unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;

  auto grid = cooperative_groups::this_grid();

  auto blockwiseScan = BlockwiseScan<cWarpSize, cWarpsPerBlock> {n, 1};
  float blockSum = blockwiseScan.scan(sTemp, values);

  if (tid == 0) dest[0] = 0;
  if (threadIdx.x == cWarpsPerBlock - 1 && blockwiseScan.getLastScannedIndex(blockIdx.x) < n)
    dest[blockwiseScan.getLastScannedIndex(blockIdx.x)] = blockSum;

  grid.sync();

  if (n > cBlockSize) {
    std::size_t step = 1;
    char additionalOffset = 0;
    float * currentSTemp = sTemp;

    do {
      step *= cBlockSize;
      currentSTemp += cWarpsPerBlock;

      // Calculate the number of points in the interval [step, n] divisible by step.
      // We make use of the fact that step is a power of 2.
      // Further, step <= n.
      //
      // E.g., in binary,
      //    n = 10110100
      //    step = 1000
      //    nPoints = n / step = 10110
      std::size_t nPoints = n / step;

      // problem: (... + additionalOffset) may go beyond the array
      BlockwiseScan<cWarpSize, cWarpsPerBlock> blockwiseScan{nPoints, step};
      float blockSum = blockwiseScan.scan(
        currentSTemp, dest + step - 1
      );
      grid.sync();

      if (threadIdx.x == cWarpsPerBlock - 1 && blockwiseScan.getLastScannedIndex(blockIdx.x) < n)
        dest[blockwiseScan.getLastScannedIndex(blockIdx.x)] = blockSum;

      grid.sync();
    } while (step <= n / cBlockSize);

    grid.sync();
/*
    for (int i = 0; i < 2 * cWarpsPerBlock; ++i) {
      dest[blockIdx.x * 2 * cWarpsPerBlock + i] = sTemp[i];
    }

    printf("Step: %lu\n", step);
*/
    do {
      std::size_t nPoints = n / step;

      float blockBaseValue = blockIdx.x > 0 ? dest[blockIdx.x * cBlockSize * step - 1] : 0;

      scanSingleStrideFillinStep<cWarpSize, cWarpsPerBlock> (
        dest + step - 1, nPoints, step, currentSTemp, blockBaseValue, dest + step - 1
      );
      grid.sync();

      step /= cBlockSize;
      currentSTemp -= cWarpsPerBlock;
      additionalOffset = !additionalOffset;
    } while (step > 1);
  }

  float blockBaseValue = blockIdx.x > 0 ? dest[blockIdx.x * cBlockSize - 1] : 0;
  scanSingleStrideFillinStep<cWarpSize, cWarpsPerBlock> (dest, n, 1, sTemp, blockBaseValue, values);
}

// assumption: n > 0
template <int cWarpSize = 32, int cWarpsPerBlock = 256 / 32>
__global__ void kernel_scan(float * dest, std::size_t n, float * values) {
  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock

  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock
  static_assert(cWarpsPerBlock <= cWarpSize, "");

  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

  unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;
  unsigned int const stride = gridDim.x * blockDim.x;
  unsigned int const wid = threadIdx.x / cWarpSize;
  unsigned int const lane = threadIdx.x % cWarpSize;

  // adjust dimension
  extern __shared__ float temp[]; 

  using MaskType = unsigned int;

  bool firstStride = true;
  std::size_t s = 0;
  if (n > stride) {
    for (; s < n - stride; s += stride) {
      scanSingleStrideSteps<cWarpSize, cWarpsPerBlock>(
        dest + s, stride, n, firstStride, temp, values + s
      );
      firstStride = false;
    }
  }
  if (n - s > 0)  {
    scanSingleStrideSteps<cWarpSize, cWarpsPerBlock>(
      dest + s, n - s, n, firstStride, temp, values + s
    );
  }
}

constexpr int cWarpSize = 32;
constexpr int cWarpsPerBlock = 1;
constexpr int cBlockSize = cWarpSize * cWarpsPerBlock;
constexpr std::size_t nSampleData = 5 * cBlockSize * cBlockSize;

std::array<float, nSampleData> sampleData;
std::array<float, nSampleData> result;
std::array<float, nSampleData> cpuResult;

int main() {
  int numBlocksPerSm = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void *)kernel_scan<cWarpSize, cWarpsPerBlock>, 
  cWarpSize * cWarpsPerBlock, 0);
  std::cerr << numBlocksPerSm << "\n";


  for (std::size_t i = 0; i < nSampleData; ++i) sampleData[i] = i;

  DeviceVector<float> d_data(sampleData);
  DeviceVector<float> d_result(nSampleData);

  auto a1 = d_result.data();
  auto a2 = nSampleData;
  auto a3 = d_data.data();
  void * kernelArgs [] = { (void *)&a1, (void *)&a2, (void *)&a3 };

  std::size_t stride = numBlocksPerSm * cWarpSize * cWarpsPerBlock;
  std::cerr << "Stride: " << stride << "\n";

  // calculate the amount of shared memory we will need
  std::size_t nSharedFloats = 0;
  for (std::size_t n = stride, step = 1; n != 0; n /= (step *= cBlockSize)) {
    nSharedFloats += cWarpsPerBlock;
  }

  std::cerr << "nSharedFloats: " << nSharedFloats << " (= cWarpsPerBlock * " << nSharedFloats / cWarpsPerBlock << ")\n";
  hipLaunchCooperativeKernel(
    (void *)kernel_scan<cWarpSize, cWarpsPerBlock>,
    dim3{numBlocksPerSm}, dim3{cWarpSize * cWarpsPerBlock},
    kernelArgs, nSharedFloats * sizeof(float)
  );
  CUDA_CHECK(hipGetLastError());

  d_result.memcpyToHost(result.data());

  constexpr auto cBlockSize = cWarpSize * cWarpsPerBlock;
  float s = 0;
  for (auto i = 0; i < nSampleData; ++i) {
    //if (!(i % cBlockSize)) s = 0; 
    s += sampleData[i]; cpuResult[i] = s;
  }
  for (int i = 0; i < 512; ++i)
  //for (int i : {256, 256 * 256 - 1, 256 * 256, 256 * 256 + 1, 256 * 256 + 256})
    std::cerr << i << " " << result[i] << " " <<
    cpuResult[i] << '\n';
  return 0;
}