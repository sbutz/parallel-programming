#include "hip/hip_runtime.h"
#include "device_vector.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// return ceil(a / b) without overflow issues
constexpr __host__ __device__ __forceinline__ std::size_t ceilDiv(std::size_t a, std::size_t b) {
  return (a / b) + ((a / b) * b > 0);
}

// return result of ceil(a / b) < c without overflow issues
constexpr __host__ __device__ __forceinline__ bool ceilDivLt(std::size_t a, std::size_t b, std::size_t c) {
  return (a - 1) / b + 1 < c;
}

// return result of a < ceil(b / c) without overflow issues
constexpr __host__ __device__ __forceinline__ bool ltCeilDiv(std::size_t a, std::size_t b, std::size_t c) {
  return (b != 0) && (b - 1) / c >= a;
}

template <int cWarpSize = 32, int cWarpsPerBlock = 256 / 32>
__device__ void scanSingleStrideStep(
  float * dest, std::size_t n, 
  std::size_t step,
  float * sTemp,
  float * values
) {
  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock
  static_assert(cWarpsPerBlock <= cWarpSize, "");

  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

  unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;
  unsigned int const stride = gridDim.x * blockDim.x;
  unsigned int const wid = threadIdx.x / cWarpSize;
  unsigned int const lane = threadIdx.x % cWarpSize;
  unsigned int const warpInGrid = tid / cWarpSize;

  using MaskType = unsigned int;
  static_assert(8 * sizeof(MaskType) == cWarpSize, "");

  if (ltCeilDiv(warpInGrid, n, cWarpSize)) {
    auto v = tid < n ? values[tid * step + (step - 1)] : 0;

    __syncwarp();

    auto const mask = MaskType{} - 1u;
    for (int w = 1; w != cWarpSize; w <<= 1) {
      float x = __shfl_up_sync(mask, v, w);
      if (lane >= w) v += x; 
    }

    if (lane == cWarpSize - 1) sTemp[wid] = v;

    __syncthreads();

    if (blockIdx.x < n / cBlockSize && wid == 0) {
      v = threadIdx.x < cWarpsPerBlock ? sTemp[threadIdx.x] : 0;
      __syncwarp();

      for (int w = 1; w < cWarpsPerBlock; w <<= 1) {
        float x = __shfl_up_sync(mask, v, w);
        if (lane >= w) v += x;
      }

      if (threadIdx.x < cWarpsPerBlock) sTemp[threadIdx.x] = v;

      if (threadIdx.x == cWarpsPerBlock - 1) {
        dest[cBlockSize * (blockIdx.x + 1) * step - 1] = v;
      }
    }

    __syncthreads();
  }
}

template <int cWarpSize = 32, int cWarpsPerBlock = 256 / 32>
__device__ void scanSingleStrideFillinStep(
  float * dest, std::size_t n, 
  std::size_t step,
  float * sTemp,
  float * values
) {
  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock
  static_assert(cWarpsPerBlock <= cWarpSize, "");

  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

  unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;
  unsigned int const stride = gridDim.x * blockDim.x;
  unsigned int const wid = threadIdx.x / cWarpSize;
  unsigned int const lane = threadIdx.x % cWarpSize;
  unsigned int const warpInGrid = tid / cWarpSize;

  using MaskType = unsigned int;
  static_assert(8 * sizeof(MaskType) == cWarpSize, "");

  if (ltCeilDiv(warpInGrid, n, cWarpSize)) {
    auto basePrevious = blockIdx.x != 0 ? dest[blockIdx.x * cBlockSize * step - 1] : 0;
    auto baseFromShared = threadIdx.x >= cWarpSize ? sTemp[threadIdx.x / cWarpSize - 1] : 0;
    auto v = tid < n ? values[tid * step] : 0;
    __syncwarp();

    auto const mask = MaskType{} - 1u;
    for (int w = 1; w < cWarpSize; w <<= 1) {
      float x = __shfl_up_sync(mask, v, w);
      if (lane >= w) v += x;
    }
    __syncwarp();

    if (tid < n) dest[tid * step] = basePrevious + baseFromShared + v;
    __syncthreads();
  }
}


template <int cWarpSize = 32, int cWarpsPerBlock = 256 / 32>
__forceinline__ __device__ void scanSingleStrideSteps(
  float * dest, std::size_t n, 
  float * sTemp,
  float * values
) {
  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

  std::size_t step = 1;
  std::size_t nn = n;
  float * currentSTemp = sTemp;
  float * vs = values;
  for (;;) {
    scanSingleStrideStep(dest, nn, step, currentSTemp, vs);
    auto grid = cooperative_groups::this_grid();
    grid.sync();

    if (nn / step == 0) break;
    step *= cBlockSize;
    nn /= step;
    currentSTemp += cWarpsPerBlock;
    vs = dest;
    //if (blockIdx.x == 0 && threadIdx.x == 0) printf("Step is %lu\n", step);
  }
  for (;;) {
    nn = n / step;
    if (blockIdx.x == 0 && threadIdx.x == 0) printf("Step is %lu\n", step);
    scanSingleStrideFillinStep(dest, nn, step, currentSTemp, values);
    auto grid = cooperative_groups::this_grid();
    grid.sync();

    if (step == 1) break;
    step /= cBlockSize;
    currentSTemp -= cWarpsPerBlock;
  }
}

// assumption: n > 0
template <int cWarpSize = 32, int cWarpsPerBlock = 256 / 32>
__global__ void kernel_scan(float * dest, std::size_t n, float * values) {
  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock

  // assert: warpSize == cWarpSize
  // assert: blockDim.x = cWarpSize * cWarpsPerBlock
  static_assert(cWarpsPerBlock <= cWarpSize, "");

  constexpr unsigned int cBlockSize = cWarpSize * cWarpsPerBlock;

  unsigned int const tid = cBlockSize * blockIdx.x + threadIdx.x;
  unsigned int const stride = gridDim.x * blockDim.x;
  unsigned int const wid = threadIdx.x / cWarpSize;
  unsigned int const lane = threadIdx.x % cWarpSize;

  // adjust dimension
  extern __shared__ float temp[]; 

  using MaskType = unsigned int;

  std::size_t s = 0;
  if (n > stride) {
    for (; s < n - stride; s += stride) {
      scanSingleStrideSteps(
        dest + s, stride, temp, values + s
      );
    }
  }
  if (n - s > 0)  {
    scanSingleStrideSteps(
      dest + s, n - s, temp, values + s
    );
  }
}

constexpr int cWarpSize = 32;
constexpr int cWarpsPerBlock = 8;
constexpr int cBlockSize = cWarpSize * cWarpsPerBlock;
constexpr std::size_t nSampleData = 5 * cBlockSize * cBlockSize;

std::array<float, nSampleData> sampleData;
std::array<float, nSampleData> result;
std::array<float, nSampleData> cpuResult;

int main() {
  int numBlocksPerSm = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void *)kernel_scan<cWarpSize, cWarpsPerBlock>, 
  cWarpSize * cWarpsPerBlock, 0);
  std::cerr << numBlocksPerSm << "\n";


  for (std::size_t i = 0; i < nSampleData; ++i) sampleData[i] = i;

  DeviceVector<float> d_data(sampleData);
  DeviceVector<float> d_result(nSampleData);

  auto a1 = d_result.data();
  auto a2 = nSampleData;
  auto a3 = d_data.data();
  void * kernelArgs [] = { (void *)&a1, (void *)&a2, (void *)&a3 };

  std::size_t stride = numBlocksPerSm * cWarpSize * cWarpsPerBlock;
  std::cerr << "Stride: " << stride << "\n";

  // calculate the amount of shared memory we will need
  std::size_t nSharedFloats = 0;
  for (std::size_t n = stride, step = 1; n != 0; n /= (step *= cBlockSize)) {
    nSharedFloats += cWarpsPerBlock;
  }

  std::cerr << "nSharedFloats: " << nSharedFloats << " (= cWarpsPerBlock * " << nSharedFloats / cWarpsPerBlock << ")\n";
  hipLaunchCooperativeKernel(
    (void *)kernel_scan<cWarpSize, cWarpsPerBlock>,
    dim3{numBlocksPerSm}, dim3{cWarpSize * cWarpsPerBlock},
    kernelArgs, nSharedFloats * sizeof(float)
  );
  CUDA_CHECK(hipGetLastError());

  d_result.memcpyToHost(result.data());

  constexpr auto cBlockSize = cWarpSize * cWarpsPerBlock;
  float s = 0;
  for (auto i = 0; i < nSampleData; ++i) {
    //if (!(i % cBlockSize)) s = 0; 
    s += sampleData[i]; cpuResult[i] = s;
  }
  for (int i = 0; i < 512; ++i)
    std::cerr << i << " " << result[i] << " " <<
    cpuResult[i] << '\n';
  return 0;
}