#include "hip/hip_runtime.h"
#include "a_bfs.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

constexpr int findNextUnvisitedCoreNaivePolicy = 1;
constexpr int findNextUnvisitedCoreSuccessivePolicy = 2;
constexpr int findNextUnvisitedCoreSuccessiveMultWarpPolicy = 3;
constexpr int findNextUnvisitedCoreSuccessiveSimplifiedPolicy = 4;

constexpr int frontierBasicPolicy = 1;
constexpr int frontierSharedPolicy = 2;
constexpr int graphTexturePolicy = 3;

// ******************************************************************************************************************************
// Auxiliary data structure for BFS
// ******************************************************************************************************************************

struct FrontierData {
    IdxType * d_frontierBuffer = nullptr;
    struct {
        IdxType * d_cntFrontier;
        IdxType * d_frontier;
    } frontiers[2];
    char currentFrontier = 0;
    IdxType maxFrontierSize;
  
    explicit FrontierData(IdxType maxFrontierSize): maxFrontierSize(maxFrontierSize) {
        // TODO: Should we malloc everything at once?
        size_t frontierBufferSize = 2 * (1 + (std::size_t)maxFrontierSize);
        CUDA_CHECK(hipMalloc(&this->d_frontierBuffer, frontierBufferSize * sizeof(IdxType)))
        CUDA_CHECK(hipMemset(this->d_frontierBuffer, 0, frontierBufferSize * sizeof(IdxType)))
        this->frontiers[0] = { this->d_frontierBuffer, this->d_frontierBuffer + 1 };
        this->frontiers[1] = { this->d_frontierBuffer + frontierBufferSize / 2, this->d_frontierBuffer + frontierBufferSize / 2 + 1 };
    }
    FrontierData(FrontierData const &) = delete;
    ~FrontierData() {
        (void)hipFree(this->d_frontierBuffer);
    }
};

static __device__ __forceinline__ void trap() {
    asm("trap;");
}

// ******************************************************************************************************************************
// FindComponent: template struct, FindComponent<FrontierPolicyKey>::findComponent will provide an interface to our BFS
// ******************************************************************************************************************************

template <int FrontierPolicyKey> struct FindComponent;

// ******************************************************************************************************************************
// FindComponent<frontierBasicPolicy>, using
//   kernel_bfs: simple BFS kernel
// ******************************************************************************************************************************

static __device__ void appendToFrontier(IdxType * cntFrontier, IdxType * frontier, IdxType maxFrontierSize, IdxType vertex) {
    IdxType old = atomicAdd(cntFrontier, 1);
    if (old >= maxFrontierSize) trap();
    frontier[old] = vertex;
}

static __global__ void kernel_bfs(
    DNeighborGraph graph,
    unsigned int * d_visited,
    unsigned int visitedTag, // must be != 0
    IdxType * cntFrontier,
    IdxType * frontier,
    IdxType * cntNewFrontier,
    IdxType * newFrontier,
    IdxType maxFrontierSize
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    auto processFrontierEntry = [&] (IdxType i) {
        IdxType vertex = frontier[i];
        IdxType incidenceListStart = graph.d_startIndices[vertex];
        IdxType incidenceListEnd = graph.d_startIndices[vertex+1];

        for (IdxType j = incidenceListStart; j < incidenceListEnd; ++j) {
            IdxType destination = graph.d_incidenceAry[j];
            unsigned int destinationVisited = d_visited[destination];
            if(destinationVisited <= 1) {
                d_visited[destination] = visitedTag;
                if (destinationVisited == 1) appendToFrontier(cntNewFrontier, newFrontier, maxFrontierSize, destination);
            }
        }
    };

    IdxType cnt = *cntFrontier;
    IdxType strideBegin = 0;
    if (cnt > stride) for (; strideBegin < cnt - stride; strideBegin += stride) processFrontierEntry(strideBegin + tid);
    if (tid < cnt - strideBegin) processFrontierEntry(strideBegin + tid);
}

template <>
struct FindComponent<frontierBasicPolicy> {
    static void findComponent(
        std::vector<std::vector<IdxType>> & visitedSteps,
        int nSm,
        IdxType * d_visited,
        FrontierData * fd,
        DNeighborGraph const * graph, IdxType startVertex, IdxType visitedTag
    ) {
        int nBlocks = 16 * nSm;
        constexpr int nThreadsPerBlock = 128;
        IdxType startValues [2] = { 1, startVertex };

        CUDA_CHECK(hipMemcpy(fd->frontiers[fd->currentFrontier].d_cntFrontier, &startValues[0], 2 * sizeof(IdxType), hipMemcpyHostToDevice))
        CUDA_CHECK(hipMemcpy(&d_visited[startVertex], &visitedTag, sizeof(IdxType), hipMemcpyHostToDevice))

        visitedSteps.push_back(std::vector<IdxType> (graph->nVertices));
        CUDA_CHECK(hipMemcpy(visitedSteps[visitedSteps.size() - 1].data(), d_visited, graph->nVertices * sizeof(IdxType), hipMemcpyDeviceToHost))
        for (;;) {
            CUDA_CHECK(hipMemset(fd->frontiers[!fd->currentFrontier].d_cntFrontier, 0, sizeof(IdxType)))

            kernel_bfs <<<nBlocks, nThreadsPerBlock>>> (
                *graph,
                d_visited,
                visitedTag,
                fd->frontiers[fd->currentFrontier].d_cntFrontier,
                fd->frontiers[fd->currentFrontier].d_frontier,
                fd->frontiers[!fd->currentFrontier].d_cntFrontier,
                fd->frontiers[!fd->currentFrontier].d_frontier,
                fd->maxFrontierSize
            );
            visitedSteps.push_back(std::vector<IdxType> (graph->nVertices));
            CUDA_CHECK(hipMemcpy(visitedSteps[visitedSteps.size() - 1].data(), d_visited, graph->nVertices * sizeof(IdxType), hipMemcpyDeviceToHost))    

            IdxType cntNewFrontier;
            CUDA_CHECK(hipMemcpy(
                &cntNewFrontier, fd->frontiers[!fd->currentFrontier].d_cntFrontier, sizeof(IdxType),
                hipMemcpyDeviceToHost
            ))
            if (!cntNewFrontier) break;
            fd->currentFrontier = !fd->currentFrontier;
        }
    }
};

// ******************************************************************************************************************************
// FindComponent<frontierSharedPolicy>, using
//   kernel_bfs_shared_frontier: BFS kernel which uses shared memory in building the frontier
// ******************************************************************************************************************************

static __device__ void appendToFrontierShared(
    IdxType * cntSharedFrontier, IdxType * sharedFrontier, unsigned int sharedFrontierSize,
    IdxType * cntFrontier, IdxType * frontier, IdxType maxFrontierSize, IdxType vertex
) {
    if (*cntSharedFrontier < sharedFrontierSize) {
        IdxType old = atomicAdd(cntSharedFrontier, 1);
        sharedFrontier[old] = vertex;
    } else {
        IdxType old = atomicAdd(cntFrontier, 1);
        if (old >= maxFrontierSize) trap();
        frontier[old] = vertex;
    }
}

static __device__ void copySharedToGlobalFrontier(
    IdxType * startPos,
    IdxType * cntSharedFrontier, IdxType * sharedFrontier, unsigned int sharedFrontierSize,
    IdxType * cntGlobalFrontier, IdxType * globalFrontier, IdxType maxFrontierSize
) {
    if (threadIdx.x == 0) *startPos = atomicAdd(cntGlobalFrontier, *cntSharedFrontier);
    __syncthreads();
    if (*startPos >= maxFrontierSize) trap();
    IdxType start = *startPos;
    IdxType stride = blockDim.x;
    for (IdxType i = threadIdx.x; i < sharedFrontierSize; i += stride) {
        globalFrontier[start + i] = sharedFrontier[i];
    }
}

static __global__ void kernel_bfs_shared_frontier(
    DNeighborGraph graph,
    unsigned int * d_visited,
    unsigned int visitedTag, // must be != 0
    IdxType * cntFrontier,
    IdxType * frontier,
    IdxType * cntNewFrontier,
    IdxType * newFrontier,
    unsigned int sharedFrontierSize,
    IdxType maxFrontierSize
) {
    __shared__ IdxType cntSharedFrontier;
    __shared__ IdxType startPos;
    extern __shared__ IdxType sharedFrontier[];

    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    if (threadIdx.x == 0) cntSharedFrontier = 0;
    __syncthreads();

    auto processFrontierEntry = [&] (IdxType i) {
        IdxType vertex = frontier[i];
        IdxType incidenceListStart = graph.d_startIndices[vertex];
        IdxType incidenceListEnd = graph.d_startIndices[vertex+1];

        for (IdxType j = incidenceListStart; j < incidenceListEnd; ++j) {
            IdxType destination = graph.d_incidenceAry[j];
            unsigned int destinationVisited = d_visited[destination];
            if(destinationVisited <= 1) {
                d_visited[destination] = visitedTag;
                if (destinationVisited == 1) appendToFrontierShared(
                    &cntSharedFrontier, sharedFrontier, sharedFrontierSize,
                    cntNewFrontier, newFrontier, maxFrontierSize,
                    destination
                );
            }
        }
    };

    IdxType cnt = *cntFrontier;
    IdxType strideBegin = 0;
    if (cnt > stride) for (; strideBegin < cnt - stride; strideBegin += stride) processFrontierEntry(strideBegin + tid);
    if (tid < cnt - strideBegin) processFrontierEntry(strideBegin + tid);

    __syncthreads();

    copySharedToGlobalFrontier(
        &startPos,
        &cntSharedFrontier, sharedFrontier, sharedFrontierSize,
        cntNewFrontier, newFrontier, maxFrontierSize
    );
}

template <>
struct FindComponent<frontierSharedPolicy> {
    static void findComponent(
        int nSm,
        IdxType * d_visited,
        FrontierData * fd,
        DNeighborGraph const * graph, IdxType startVertex, IdxType visitedTag
    ) {
        int nBlocks = 2 * nSm;
        unsigned int sharedFrontierSize = (1u << 13) / (nBlocks / nSm); // 8 * 1024 values -> 4 * 8 * 1024 Bytes = 32 kiB
        constexpr int nThreadsPerBlock = 512;

        IdxType startValues [2] = { 1, startVertex };

        CUDA_CHECK(hipMemcpy(fd->frontiers[fd->currentFrontier].d_cntFrontier, &startValues, 2 * sizeof(IdxType), hipMemcpyHostToDevice))
        CUDA_CHECK(hipMemcpy(&d_visited[startVertex], &visitedTag, sizeof(IdxType), hipMemcpyHostToDevice))
        for (;;) {
            CUDA_CHECK(hipMemset(fd->frontiers[!fd->currentFrontier].d_cntFrontier, 0, sizeof(IdxType)))
            kernel_bfs_shared_frontier <<<nBlocks, nThreadsPerBlock, sharedFrontierSize * sizeof(IdxType)>>> (
                *graph,
                d_visited,
                visitedTag,
                fd->frontiers[fd->currentFrontier].d_cntFrontier,
                fd->frontiers[fd->currentFrontier].d_frontier,
                fd->frontiers[!fd->currentFrontier].d_cntFrontier,
                fd->frontiers[!fd->currentFrontier].d_frontier,
                sharedFrontierSize,
                fd->maxFrontierSize
            );

            IdxType cntNewFrontier;
            CUDA_CHECK(hipMemcpy(
                &cntNewFrontier, fd->frontiers[!fd->currentFrontier].d_cntFrontier, sizeof(IdxType),
                hipMemcpyDeviceToHost
            ))
            if (!cntNewFrontier) break;
            fd->currentFrontier = !fd->currentFrontier;
        }
    }
};

// ******************************************************************************************************************************
// markCore: helper function for initializing clusters array by marking core elements
// ******************************************************************************************************************************

static __global__ void kernel_markCoreUnvisited(
    IdxType * d_visited,
    IdxType * d_d_startIndices,
    IdxType nVertices
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nVertices) {
        if (d_d_startIndices[tid + 1] - d_d_startIndices[tid] != 0) {
            d_visited[tid] = 1;
        }
    }    
}

static void markCoreUnvisited(IdxType * d_visited, DNeighborGraph const * graph) {
    constexpr int nThreadsPerBlock = 128;
    kernel_markCoreUnvisited <<<
        dim3((graph->nVertices + nThreadsPerBlock - 1) / nThreadsPerBlock),
        dim3(nThreadsPerBlock)    
    >>> (
        d_visited,
        graph->d_startIndices,
        graph->nVertices
    );
    CUDA_CHECK(hipGetLastError())
}

// ******************************************************************************************************************************
// FindNextUnvisitedCore:
//   template struct, FindNextUnvisitedCore<FrontierPolicyKey>::findNextUnvisitedCore finds next unvisited (core) node
// ******************************************************************************************************************************

template <int FindNextUnvisitedCorePolicyKey>
struct FindNextUnvisitedCore {
    struct Result {
        bool wasFound;
        IdxType idx;
    };
    static Result findNextUnvisitedCore(
        IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
    );
};

// ******************************************************************************************************************************
// FindNextUnvisitedCore<findNextUnvisitedCoreNaivePolicy>
//   naive, but simple way of finding next unvisited (core) node
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedCoreNaive(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nVertices) {
        if (d_visited[tid] == 1) {
            outBuffer[0] = 1; // true
            outBuffer[1] = tid;
        }
    }    
}

template <>
auto FindNextUnvisitedCore<findNextUnvisitedCoreNaivePolicy>::findNextUnvisitedCore(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisitedCore<findNextUnvisitedCoreNaivePolicy>::Result {
    constexpr int nThreadsPerBlock = 128;
    CUDA_CHECK(hipMemset(d_resultBuffer, 0, 2 * sizeof(IdxType)))

    kernel_findUnvisitedCoreNaive <<<
        (nVertices + nThreadsPerBlock - 1) / nThreadsPerBlock,
        nThreadsPerBlock
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);

    IdxType localBuffer [2];
    CUDA_CHECK(hipMemcpy(localBuffer, d_resultBuffer, 2 * sizeof(IdxType), hipMemcpyDeviceToHost))

    return {!!localBuffer[0], localBuffer[1]};
}

// ******************************************************************************************************************************
// FindNextUnvisitedCore<findNextUnvisitedCoreSuccessivePolicy>
//   start over where you stopped, rather than at the beginning
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedCoreSuccessive(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    constexpr int logWrp = 5;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;
    unsigned int tid = threadIdx.x;
    unsigned int idx = (startPos & ~(wrp - 1)) + tid;

    IdxType contribution;
    for (;;) {
        contribution = idx < startPos || idx >= nVertices || d_visited[idx] != 1?
            maxIdxType : idx;

        #pragma unroll
        for (int delta = 1; delta < wrp; delta <<= 1) {
            auto other = __shfl_down_sync(0xffffffff, contribution, delta);
            if (other < contribution) contribution = other;
        }

        contribution = __shfl_sync(0xffffffff, contribution, 0);

        if ((idx >> logWrp) == (nVertices >> logWrp) || contribution != maxIdxType) break;

        idx += wrp;
    };

    if (tid == 0) *outBuffer = contribution;
}

template <>
auto FindNextUnvisitedCore<findNextUnvisitedCoreSuccessivePolicy>::findNextUnvisitedCore(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisitedCore<findNextUnvisitedCoreSuccessivePolicy>::Result {
    constexpr int nThreadsPerBlock = 32;
    constexpr int blocks = 1;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    IdxType localBuffer;
    kernel_findUnvisitedCoreSuccessive <<<
        blocks, nThreadsPerBlock
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);

    CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

    return {localBuffer != maxIdxType, localBuffer};
}

// ******************************************************************************************************************************
// FindNextUnvisitedCore<findNextUnvisitedCoreSuccessiveSimplifiedPolicy>
//   __ballot_sync rather than __shfl_sync
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedCoreSuccessiveSimplified(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    IdxType result = (IdxType)-1;
    for (IdxType strideIdx = startPos / wrp; strideIdx <= ((nVertices - 1) / wrp); ++strideIdx) {
        IdxType idx = strideIdx * wrp + threadIdx.x;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < nVertices && d_visited[idx] == 1);
        if (unvisitedMask != 0) {
            result = strideIdx * wrp + __ffs(unvisitedMask) - 1;
            break;
        }
    }
    if (threadIdx.x == 0) *outBuffer = result;
}

template <>
auto FindNextUnvisitedCore<findNextUnvisitedCoreSuccessiveSimplifiedPolicy>::findNextUnvisitedCore(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisitedCore<findNextUnvisitedCoreSuccessiveSimplifiedPolicy>::Result {
    constexpr int nThreadsPerBlock = 32;
    constexpr int blocks = 1;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    if (startIdx >= nVertices) return Result{false, 0};

    IdxType localBuffer;
    kernel_findUnvisitedCoreSuccessiveSimplified <<<
        blocks, nThreadsPerBlock
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);

    CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

    return Result{localBuffer != maxIdxType, localBuffer};
}

// ******************************************************************************************************************************
// FindNextUnvisitedCore<findNextUnvisitedCoreSuccessiveMultWarpPolicy>
//   __ballot_sync rather than __shfl_sync, employ several warps
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedCoreSuccessiveMultWarp(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    constexpr unsigned int stride = 2 * wrp;
    constexpr unsigned int strideStartMask = ~(stride - 1);
    constexpr int warpsPerBlock = stride / wrp;

    __shared__ unsigned int contributions[warpsPerBlock];

    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    //unsigned int stride = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int wid = threadIdx.x / wrp;
    unsigned int lane = threadIdx.x % wrp;

    IdxType strideStartIdx = (startPos & ~(wrp - 1));

    IdxType contribution;
    for (;;) {
        // ! TODO: this may overflow
        IdxType idx = strideStartIdx + tid;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < nVertices && d_visited[idx] == 1);

        if (lane == 0) contributions[wid] = unvisitedMask ? strideStartIdx + wrp * wid + __ffs(unvisitedMask) - 1 : maxIdxType;

        __syncthreads();

        if (wid == 0) {
            contribution = tid < warpsPerBlock ? contributions[tid] : maxIdxType;

            #pragma unroll
            for (int delta = 1; delta < warpsPerBlock; delta <<= 1) {
                auto other = __shfl_down_sync(0xffffffff, contribution, delta);
                if (other < contribution) contribution = other;
            }

            if (tid == 0) contributions[0] = contribution;
        }

        __syncthreads();

        contribution = contributions[0];

        if (strideStartIdx >= (nVertices & strideStartMask) || contribution != maxIdxType) break;

        strideStartIdx += stride;
    };

    if (tid == 0) *outBuffer = contribution;
}

template <>
auto FindNextUnvisitedCore<findNextUnvisitedCoreSuccessiveMultWarpPolicy>::findNextUnvisitedCore(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisitedCore<findNextUnvisitedCoreSuccessiveMultWarpPolicy>::Result {
    constexpr int nThreadsPerBlock = 2 * 32;
    constexpr int blocks = 1;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    IdxType localBuffer;
    kernel_findUnvisitedCoreSuccessiveMultWarp <<<
        dim3(blocks), dim3(nThreadsPerBlock)
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);
    CUDA_CHECK(hipGetLastError())

    CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

        struct Result {
            bool wasFound;
            IdxType idx;
        };
    return {localBuffer != maxIdxType, localBuffer};
}

// ******************************************************************************************************************************
// findAllComponents: find all the clusters
// ******************************************************************************************************************************

template <int FindNextUnvisitedCorePolicyKey, int FrontierPolicyKey>
static void findAllComponents(
    int nSm,
    IdxType * d_visited,
    FindComponentsProfilingData * profile,
    DNeighborGraph const * graph,
    std::vector<std::vector<IdxType>> & visitedSteps
) {
    FrontierData fd{graph->lenIncidenceAry};

    IdxType nextFreeTag = 2;
    ManagedDeviceArray<IdxType> d_resultBuffer {2};

    CUDA_CHECK(hipMemset(d_visited, 0, graph->nVertices * sizeof(IdxType)))

    profile->timeMarkCoreUnvisited = runAndMeasureCuda(markCoreUnvisited, d_visited, graph);
    profile->timeFindComponents = runAndMeasureCuda([&]{
        IdxType nIterations = 0;
        IdxType startIdx = 0;
        for (;;) {
            auto nextUnvisitedCore = FindNextUnvisitedCore<FindNextUnvisitedCorePolicyKey>::findNextUnvisitedCore(
                d_resultBuffer.ptr(), d_visited, graph->nVertices, startIdx
            );
            if (!nextUnvisitedCore.wasFound) break;
            FindComponent<FrontierPolicyKey>::findComponent(visitedSteps, nSm, d_visited, &fd, graph, nextUnvisitedCore.idx, nextFreeTag);
            startIdx = nextUnvisitedCore.idx + 1;
            ++nextFreeTag;
            ++nIterations;
        }
        CUDA_CHECK(hipGetLastError())
    });
}
