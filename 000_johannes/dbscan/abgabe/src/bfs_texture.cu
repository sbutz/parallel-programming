#include "hip/hip_runtime.h"
#include "bfs.h"
#include "cuda_helpers.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

// ******************************************************************************************************************************
// Auxiliary data structure for BFS
// ******************************************************************************************************************************

struct FrontierData {
    IdxType * d_frontierBuffer = nullptr;
    struct {
        IdxType * d_cntFrontier;
        IdxType * d_frontier;
    } frontiers[2];
    char currentFrontier = 0;
  
    FrontierData(size_t maxFrontierSize) {
        // TODO: Should we malloc everything at once?
        size_t frontierBufferSize = 2 * (1 + (std::size_t)maxFrontierSize);
        CUDA_CHECK(hipMalloc(&this->d_frontierBuffer, frontierBufferSize * sizeof(IdxType)))
        CUDA_CHECK(hipMemset(this->d_frontierBuffer, 0, frontierBufferSize * sizeof(IdxType)))
        this->frontiers[0] = { this->d_frontierBuffer, this->d_frontierBuffer + 1 };
        this->frontiers[1] = { this->d_frontierBuffer + frontierBufferSize / 2, this->d_frontierBuffer + frontierBufferSize / 2 + 1 };
    }
    FrontierData(FrontierData const &) = delete;
    ~FrontierData() {
        (void)hipFree(this->d_frontierBuffer);
    }
};

// ******************************************************************************************************************************
// FindComponent: template struct, FindComponent<FrontierPolicyKey>::findComponent will provide an interface to our BFS
// ******************************************************************************************************************************

template <int FrontierPolicyKey> struct FindComponent;

// ******************************************************************************************************************************
// FindComponent<graphTexturePolicy>, using
//   kernel_bfs_texture: kernel relying on texture memory for incidence lists
// ******************************************************************************************************************************

static __device__ void appendToFrontier(IdxType * cntFrontier, IdxType * frontier, IdxType vertex) {
    IdxType old = atomicAdd(cntFrontier, 1);
    frontier[old] = vertex;
}

texture<IdxType, 1, hipReadModeElementType> startIndicesTexture;

static __global__ void kernel_bfs_texture(
    DNeighborGraph graph,
    unsigned int * d_visited,
    unsigned int visitedTag, // must be != 0
    IdxType * cntFrontier,
    IdxType * frontier,
    IdxType * cntNewFrontier,
    IdxType * newFrontier
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    auto processFrontierEntry = [&] (IdxType i) {
        IdxType vertex = frontier[i];
        IdxType incidenceListStart = tex1D(startIndicesTexture, vertex);
        IdxType incidenceListEnd = tex1D(startIndicesTexture, vertex + 1);

        for (IdxType j = incidenceListStart; j < incidenceListEnd; ++j) {
            IdxType destination = graph.d_incidenceAry[j];
            unsigned int destinationVisited = d_visited[destination];
            if(destinationVisited <= 1) {
                d_visited[destination] = visitedTag;
                if (destinationVisited == 0) appendToFrontier(cntNewFrontier, newFrontier, destination);
            }
        }
    };

    IdxType cnt = *cntFrontier;
    IdxType strideBegin = 0;
    if (cnt > stride) for (; strideBegin < cnt - stride; strideBegin += stride) processFrontierEntry(strideBegin + tid);
    if (tid < cnt - strideBegin) processFrontierEntry(strideBegin + tid);
}

template <>
struct FindComponent<graphTexturePolicy> {
    static void findComponent(
        int nSm,
        IdxType * d_visited,
        FrontierData * fd,
        DNeighborGraph const * graph, IdxType startVertex, IdxType visitedTag
    ) {
        int nBlocks = 16 * nSm;
        constexpr int nThreadsPerBlock = 128;
        IdxType startValues [2] = { 1, startVertex };

        CUDA_CHECK(hipMemcpy(fd->frontiers[fd->currentFrontier].d_cntFrontier, &startValues[0], 2 * sizeof(IdxType), hipMemcpyHostToDevice))
        CUDA_CHECK(hipMemcpy(&d_visited[startVertex], &visitedTag, sizeof(IdxType), hipMemcpyHostToDevice))

        for (;;) {
            CUDA_CHECK(hipMemset(fd->frontiers[!fd->currentFrontier].d_cntFrontier, 0, sizeof(IdxType)))

            kernel_bfs_texture <<<nBlocks, nThreadsPerBlock>>> (
                *graph,
                d_visited,
                visitedTag,
                fd->frontiers[fd->currentFrontier].d_cntFrontier,
                fd->frontiers[fd->currentFrontier].d_frontier,
                fd->frontiers[!fd->currentFrontier].d_cntFrontier,
                fd->frontiers[!fd->currentFrontier].d_frontier
            );

            IdxType cntNewFrontier;
            CUDA_CHECK(hipMemcpy(
                &cntNewFrontier, fd->frontiers[!fd->currentFrontier].d_cntFrontier, sizeof(IdxType),
                hipMemcpyDeviceToHost
            ))
            if (!cntNewFrontier) break;
            fd->currentFrontier = !fd->currentFrontier;
        }
    }
};

// ******************************************************************************************************************************
// markNonCore: helper function for initializing clusters array by marking non-core elements
// ******************************************************************************************************************************

static __global__ void kernel_markNonCore(
    IdxType * d_visited,
    IdxType * d_d_startIndices,
    IdxType nVertices
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nVertices) {
        if (d_d_startIndices[tid + 1] - d_d_startIndices[tid] == 0) {
            d_visited[tid] = 1;
        }
    }    
}

static void markNonCore(IdxType * d_visited, DNeighborGraph const * graph) {
    constexpr int nThreadsPerBlock = 128;
    kernel_markNonCore <<<
        dim3((graph->nVertices + nThreadsPerBlock - 1) / nThreadsPerBlock),
        dim3(nThreadsPerBlock)    
    >>> (
        d_visited,
        graph->d_startIndices,
        graph->nVertices
    );
    CUDA_CHECK(hipGetLastError())
}

// ******************************************************************************************************************************
// FindNextUnvisited:
//   template struct, FindNextUnvisited<FrontierPolicyKey>::findNextUnvisited finds next unvisited (core) node
// ******************************************************************************************************************************

template <int FindNextUnvisitedPolicyKey>
struct FindNextUnvisited {
    struct Result {
        bool wasFound;
        IdxType idx;
    };
    static Result findNextUnvisited(
        IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
    );
};

// ******************************************************************************************************************************
// FindNextUnvisited<findNextUnvisitedNaivePolicy>
//   naive, but simple way of finding next unvisited (core) node
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedNaive(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // TODO: This is bad. We should start at startPos and finish early when vertex has been found.
    if (tid < nVertices) {
        if (!d_visited[tid]) {
            outBuffer[0] = 1; // true
            outBuffer[1] = tid;
        }
    }    
}

template <>
auto FindNextUnvisited<findNextUnvisitedNaivePolicy>::findNextUnvisited(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisited<findNextUnvisitedNaivePolicy>::Result {
    constexpr int nThreadsPerBlock = 128;
    CUDA_CHECK(hipMemset(d_resultBuffer, 0, 2 * sizeof(IdxType)))

    kernel_findUnvisitedNaive <<<
        (nVertices + nThreadsPerBlock - 1) / nThreadsPerBlock,
        nThreadsPerBlock
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);

    IdxType localBuffer [2];
    CUDA_CHECK(hipMemcpy(localBuffer, d_resultBuffer, 2 * sizeof(IdxType), hipMemcpyDeviceToHost))

    return {!!localBuffer[0], localBuffer[1]};
}

// ******************************************************************************************************************************
// FindNextUnvisited<findNextUnvisitedSuccessivePolicy>
//   start over where you stopped, rather than at the beginning
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedSuccessive(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    constexpr int logWrp = 5;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;
    unsigned int tid = threadIdx.x;
    unsigned int idx = (startPos & ~(wrp - 1)) + tid;

    IdxType contribution;
    for (;;) {
        contribution = idx < startPos || idx >= nVertices || !!d_visited[idx] ?
            maxIdxType : idx;

        #pragma unroll
        for (int delta = 1; delta < wrp; delta <<= 1) {
            auto other = __shfl_down_sync(0xffffffff, contribution, delta);
            if (other < contribution) contribution = other;
        }

        contribution = __shfl_sync(0xffffffff, contribution, 0);

        if ((idx >> logWrp) == (nVertices >> logWrp) || contribution != maxIdxType) break;

        idx += wrp;
    };

    if (tid == 0) *outBuffer = contribution;
}

template <>
auto FindNextUnvisited<findNextUnvisitedSuccessivePolicy>::findNextUnvisited(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisited<findNextUnvisitedSuccessivePolicy>::Result {
    constexpr int nThreadsPerBlock = 32;
    constexpr int blocks = 1;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    IdxType localBuffer;
    kernel_findUnvisitedSuccessive <<<
        blocks, nThreadsPerBlock
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);

    CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

    return {localBuffer != maxIdxType, localBuffer};
}

// ******************************************************************************************************************************
// FindNextUnvisited<findNextUnvisitedSuccessiveSimplifiedPolicy>
//   __ballot_sync rather than __shfl_sync
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedSuccessiveSimplified(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    IdxType result = (IdxType)-1;
    for (IdxType strideIdx = startPos / wrp; strideIdx <= ((nVertices - 1) / wrp); ++strideIdx) {
        IdxType idx = strideIdx * wrp + threadIdx.x;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < nVertices && !d_visited[idx]);
        if (unvisitedMask != 0) {
            result = strideIdx * wrp + __ffs(unvisitedMask) - 1;
            break;
        }
    }
    if (threadIdx.x == 0) *outBuffer = result;
}

template <>
auto FindNextUnvisited<findNextUnvisitedSuccessiveSimplifiedPolicy>::findNextUnvisited(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisited<findNextUnvisitedSuccessiveSimplifiedPolicy>::Result {
    constexpr int nThreadsPerBlock = 32;
    constexpr int blocks = 1;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    if (startIdx >= nVertices) return Result{false, 0};

    IdxType localBuffer;
    kernel_findUnvisitedSuccessiveSimplified <<<
        blocks, nThreadsPerBlock
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);

    CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

    return Result{localBuffer != maxIdxType, localBuffer};
}

// ******************************************************************************************************************************
// FindNextUnvisited<findNextUnvisitedSuccessiveMultWarpPolicy>
//   __ballot_sync rather than __shfl_sync, employ several warps
// ******************************************************************************************************************************

static __global__ void kernel_findUnvisitedSuccessiveMultWarp(
    IdxType * outBuffer,
    IdxType * d_visited,
    IdxType nVertices,
    IdxType startPos
) {
    constexpr unsigned int wrp = 32;
    constexpr unsigned int stride = 2 * wrp;
    constexpr unsigned int strideStartMask = ~(stride - 1);
    constexpr int warpsPerBlock = stride / wrp;

    __shared__ unsigned int contributions[warpsPerBlock];

    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    //unsigned int stride = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int wid = threadIdx.x / wrp;
    unsigned int lane = threadIdx.x % wrp;

    IdxType strideStartIdx = (startPos & ~(wrp - 1));

    IdxType contribution;
    for (;;) {
        // ! TODO: this may overflow
        IdxType idx = strideStartIdx + tid;
        int unvisitedMask = __ballot_sync(0xffffffff, idx >= startPos && idx < nVertices && !d_visited[idx]);

        if (lane == 0) contributions[wid] = unvisitedMask ? strideStartIdx + wrp * wid + __ffs(unvisitedMask) - 1 : maxIdxType;

        __syncthreads();

        if (wid == 0) {
            contribution = tid < warpsPerBlock ? contributions[tid] : maxIdxType;

            #pragma unroll
            for (int delta = 1; delta < warpsPerBlock; delta <<= 1) {
                auto other = __shfl_down_sync(0xffffffff, contribution, delta);
                if (other < contribution) contribution = other;
            }

            if (tid == 0) contributions[0] = contribution;
        }

        __syncthreads();

        contribution = contributions[0];

        if (strideStartIdx >= (nVertices & strideStartMask) || contribution != maxIdxType) break;

        strideStartIdx += stride;
    };

    if (tid == 0) *outBuffer = contribution;
}

template <>
auto FindNextUnvisited<findNextUnvisitedSuccessiveMultWarpPolicy>::findNextUnvisited(
    IdxType * d_resultBuffer, IdxType * d_visited, IdxType nVertices, IdxType startIdx
) -> FindNextUnvisited<findNextUnvisitedSuccessiveMultWarpPolicy>::Result {
    constexpr int nThreadsPerBlock = 2 * 32;
    constexpr int blocks = 1;
    constexpr IdxType maxIdxType = (IdxType)0 - (IdxType)1;

    IdxType localBuffer;
    kernel_findUnvisitedSuccessiveMultWarp <<<
        dim3(blocks), dim3(nThreadsPerBlock)
    >>> (d_resultBuffer, d_visited, nVertices, startIdx);
    CUDA_CHECK(hipGetLastError())

    CUDA_CHECK(hipMemcpy(&localBuffer, d_resultBuffer, sizeof(IdxType), hipMemcpyDeviceToHost))

        struct Result {
            bool wasFound;
            IdxType idx;
        };
    return {localBuffer != maxIdxType, localBuffer};
}

// ******************************************************************************************************************************
// findAllComponents: find all the clusters
// ******************************************************************************************************************************

template <int FindNextUnvisitedPolicyKey, int FrontierPolicyKey>
void findAllComponents(
    int nSm,
    IdxType * d_visited,
    FindComponentsProfilingData * profile,
    DNeighborGraph const * graph
) {
    FrontierData fd{graph->lenIncidenceAry};

    IdxType nextFreeTag = 2;
    ManagedDeviceArray<IdxType> d_resultBuffer {2};

    CUDA_CHECK(hipMemset(d_visited, 0, graph->nVertices * sizeof(IdxType)))

    hipArray * texArray = nullptr;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<IdxType> ();
    CUDA_CHECK(hipMallocArray(&texArray, &channelDesc, graph->nVertices + 1))
    CUDA_CHECK(hipMemcpy2DToArray(
        texArray, 0, 0, graph->d_startIndices,
        graph->nVertices + 1, graph->nVertices + 1, 1,
        hipMemcpyDeviceToDevice
    ))
    CUDA_CHECK(hipBindTextureToArray(startIndicesTexture, texArray))

    profile->timeMarkNonCore = runAndMeasureCuda(markNonCore, d_visited, graph);
    profile->timeFindComponents = runAndMeasureCuda([&]{
        IdxType nIterations = 0;
        IdxType startIdx = 1;
        for (;;) {
            auto nextUnvisited = FindNextUnvisited<FindNextUnvisitedPolicyKey>::findNextUnvisited(
                d_resultBuffer.ptr(), d_visited, graph->nVertices, startIdx
            );
            if (!nextUnvisited.wasFound) break;
            FindComponent<graphTexturePolicy>::findComponent(nSm, d_visited, &fd, graph, nextUnvisited.idx, nextFreeTag);
            startIdx = nextUnvisited.idx + 1;
            ++nextFreeTag;
            ++nIterations;
        }
    });
}

static void forceInstantiation() __attribute__ ((unused));
static void forceInstantiation() {
    findAllComponents<findNextUnvisitedSuccessivePolicy, frontierSharedPolicy> (0, nullptr, nullptr, nullptr);
}
