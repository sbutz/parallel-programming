#include "hip/hip_runtime.h"
#include "c_cluster_expansion.h"
#include "c_types.h"
#include "cuda_helpers.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

DPoints copyPointsToDevice(float const * x, float const * y, IdxType n) {
  float * d_x, * d_y;
  CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)))
  CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)))
  CUDA_CHECK(hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice))
  CUDA_CHECK(hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice))
  return { n, d_x, d_y };
}

static __device__ IdxType unionizeClusters(
  IdxType * clusters,
  IdxType cluster1,
  IdxType cluster2
) {
  IdxType grandchild, child, parentOffset, top2, top1;

  // the following seems to save some time
  if (cluster1 == cluster2) return cluster1;

  top1 = cluster1; // we initially assume cluster1 is the top node
  child = cluster2;
  for (;;) {
    parentOffset = clusters[child];
    if (parentOffset) {
      grandchild = child;
      child += parentOffset;
      parentOffset = clusters[child];
      while (parentOffset) {
        (void)atomicCAS(&clusters[grandchild], child - grandchild, child + parentOffset - grandchild);
        grandchild = child;
        child += parentOffset;
        parentOffset = clusters[child];
      }
    }
    top2 = child;

    if (top1 == top2) break; // necessary?
    if (top1 > top2) { IdxType tmp = top2; top2 = top1; top1 = tmp; }

    IdxType old = atomicCAS(&clusters[top2], 0, top1 - top2);
    if (!old) break;
    child = top2;
  }
  return top1;
}

static __device__ __forceinline__ unsigned int laneId() {
  unsigned ret;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(ret));
  return ret;
}

struct LargeStridePolicy {
  static __device__ auto tryAppendToNeighborBuffer(
    IdxType * s_neighborBuffer,
    IdxType * s_neighborCount,
    IdxType maxLength,
    IdxType pointIdx
  ) {
    int lane = laneId(); // or threadIdx.x & 0x1f
    unsigned int neighborMask = __ballot_sync(__activemask(), 1);
    int leader = __ffs(neighborMask) - 1;
    int nNeighbors = __popc(neighborMask);
    int oldNeighborCount;
    if (lane == leader) oldNeighborCount = atomicAdd(s_neighborCount, nNeighbors);
    oldNeighborCount = __shfl_sync(neighborMask, oldNeighborCount, leader);
    int h = oldNeighborCount + __popc(neighborMask & ((1u << lane) - 1));

    bool shouldAppend = h < maxLength;
    if (shouldAppend) s_neighborBuffer[h] = pointIdx;

    struct Result {
      bool wasAppended;
      bool maxLengthReached;
    }; 
    return Result { shouldAppend, oldNeighborCount + nNeighbors >= maxLength };
  }
};

static constexpr __host__ __device__ __forceinline__ IdxType dhi_max(IdxType a, IdxType b) {
  return a > b ? a : b;
}

static __device__ IdxType unionizeWithinThreadGroup(
  volatile IdxType * s_interWarpUnionize,
  IdxType * clusters,
  IdxType myClusterId
) {
  // unionize within every warp
  for (unsigned int i = 1; i < 32; i <<= 1) {
    IdxType otherClusterId = __shfl_down_sync(0xffffffff, myClusterId, i);
    if (!(laneId() & ((i << 1) - 1))) myClusterId = unionizeClusters(clusters, myClusterId, otherClusterId);
  }
  myClusterId = __shfl_sync(0xffffffff, myClusterId, 0);

  __syncthreads();

  // unionize among warps within thread group
  unsigned int nValuesToUnionize = (blockDim.x + 31) / 32;
  if (nValuesToUnionize > 1) {
    int lane = laneId();
    int wid = threadIdx.x / 32;
    if (lane == 0) s_interWarpUnionize[threadIdx.x / 32] = myClusterId;
    while ((wid + 1) * 32 <= nValuesToUnionize) {
      __syncthreads();
      IdxType myValue = threadIdx.x < nValuesToUnionize ? s_interWarpUnionize[threadIdx.x] : (IdxType)-1;
      unsigned int limit = nValuesToUnionize - wid * 32;
      if (limit > 32) limit = 32;
      for (unsigned int i = 1; i < limit; i <<= 1) {
        IdxType otherValue = __shfl_down_sync(0xffffffff, myValue, i);
        if (
          (otherValue + 1) &&
          !(laneId() & ((i << 1) - 1))
        ) myValue = unionizeClusters(clusters, myValue, otherValue);
      }
      nValuesToUnionize = (nValuesToUnionize + 31) / 32;
      if (laneId() == 0) s_interWarpUnionize[threadIdx.x / 32] == myValue;
      if (nValuesToUnionize == 1) break;
    }
    __syncthreads();
    myClusterId = s_interWarpUnionize[0];
  }

  return myClusterId;
}

static __global__ void kernel_handleCollisions(
  unsigned int * collisionMatrix,
  IdxType * clusters,
  bool * coreMarkers,
  IdxType n,
  IdxType beginStep,
  IdxType nThreadGroupsTotal
) {
  unsigned int collisionPitch = (nThreadGroupsTotal + 31) / 32;
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int nRequiredThreads = nThreadGroupsTotal * nThreadGroupsTotal;
  if (tid < nRequiredThreads) {
    unsigned int r = tid / nThreadGroupsTotal;
    unsigned int c = tid % nThreadGroupsTotal;

    bool collisionRToC = collisionMatrix[r * collisionPitch + c / 32] & (1u << (c % 32));
    bool collisionCToR = collisionMatrix[c * collisionPitch + r / 32] & (1u << (r % 32));
    if (c < r && collisionRToC && collisionCToR) {
      // both points are core
      (void)unionizeClusters(clusters, beginStep + r, beginStep + c);
    } else if (c < r && collisionRToC) {
      // r is core
      clusters[beginStep + c] = r - c;
    } else if (c < r && collisionCToR) {
      // c is core
      clusters[beginStep + r] = c - r;
    }
  }
}

static __device__ void sharedMemZero(
  char * s_mem, IdxType nBytes
) {
  // zeroing one byte per thread is faster than one unsigned int per thread
  //   -- reason unclear, but may be related to memory bank conflicts
  IdxType strideStart = 0;
  IdxType myOffset = blockDim.x * threadIdx.y + threadIdx.x;
  while (nBytes - strideStart > blockDim.x * blockDim.y) {
    s_mem [strideStart + myOffset] = 0;
    strideStart += blockDim.x * blockDim.y;
  }
  if (myOffset < nBytes - strideStart) s_mem [strideStart + myOffset] = 0;

}

static __device__ IdxType processPoints(
  bool * coreMarkers, IdxType * clusters, bool * s_collisions,
  IdxType * s_neighborBuffer, IdxType * s_neighborCount,
  float const * xs, float const * ys, IdxType n,
  IdxType coreThreshold, float rsq,
  IdxType beginStep, IdxType endStep,
  IdxType ourPointIdx, IdxType myClusterId
) {
  unsigned int stride = blockDim.x;

  auto markAsCandidate = [&] (IdxType pointIdx) {
    if (pointIdx < beginStep) {
      if (coreMarkers[pointIdx]) {
        myClusterId = unionizeClusters(clusters, pointIdx, myClusterId);
      } else {
        clusters[pointIdx] = myClusterId - pointIdx;
      }
    } else if (pointIdx < endStep) {
      s_collisions[pointIdx - beginStep] = true;
    } else {
      clusters[pointIdx] = myClusterId - pointIdx;
    }
  };

  float x = xs[ourPointIdx], y = ys[ourPointIdx];
  bool isDefinitelyCore = false;

  auto processObject = [&] (IdxType pointIdx) {
    float dx = xs[pointIdx] - x;
    float dy = ys[pointIdx] - y;
    bool isNeighbor = dx * dx + dy * dy <= rsq;
    if (isNeighbor) {
      bool handleImmediately = isDefinitelyCore;
      if (!handleImmediately) {
        auto r = LargeStridePolicy::tryAppendToNeighborBuffer(s_neighborBuffer, s_neighborCount, coreThreshold, pointIdx);
        handleImmediately = !r.wasAppended;
        isDefinitelyCore = r.maxLengthReached;
      }
      if (handleImmediately) markAsCandidate(pointIdx);
    }
  };

  IdxType strideIdx = 0;
  for (; strideIdx < (n - 1) / stride; ++strideIdx) processObject(strideIdx * stride + threadIdx.x);
  if (threadIdx.x < n - strideIdx * stride) processObject(strideIdx * stride + threadIdx.x);
    
  __syncthreads();

  if (*s_neighborCount >= coreThreshold) {
    for (int i = threadIdx.x; i < coreThreshold; i += stride) {
      markAsCandidate(s_neighborBuffer[i]);
    }
  }

  return myClusterId;
}

static __device__ void writeCollisions(
  unsigned int * collisionMatrix,
  bool * s_collisions,
  IdxType nThreadGroupsTotal
) {
  unsigned int stride = blockDim.x;
  unsigned int threadGroupIdx = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int collisionPitch = (nThreadGroupsTotal + 31) / 32;

  unsigned int strideBegin = 0;

  auto assembleMask = [&] (unsigned int m) {
    #pragma unroll
    for (int j = 1; j < 32; j <<= 1) {
      m |= __shfl_down_sync(0xffffffff, m, j) << j;
    }
    return m;
  };

  if (stride < nThreadGroupsTotal) {
    for (; strideBegin < nThreadGroupsTotal - stride; strideBegin += stride) {
      unsigned int collisionMask = assembleMask(s_collisions[strideBegin + threadIdx.x]);
      if (laneId() == 0) {
        collisionMatrix[collisionPitch * threadGroupIdx + (strideBegin + threadIdx.x) / 32] = collisionMask;
      }
    }
  }
  unsigned int collisionMask = assembleMask(strideBegin + threadIdx.x < nThreadGroupsTotal ? s_collisions[strideBegin + threadIdx.x] : 0);
  if (laneId() == 0 && strideBegin + threadIdx.x < nThreadGroupsTotal) collisionMatrix[collisionPitch * threadGroupIdx + (strideBegin + threadIdx.x) / 32] = collisionMask;
}

static __global__ void kernel_clusterExpansion(
  IdxType * clusters, bool * coreMarkers,
  float const * xs, float const * ys, IdxType n,
  IdxType beginStep,
  unsigned int * collisionMatrix,
  IdxType coreThreshold, float rsq
) {
  unsigned int stride = blockDim.x; // blockDim.x must be a multiple of 32
  unsigned int nThreadGroupsPerBlock = blockDim.y;
  unsigned int nBlocks = gridDim.y;
  unsigned int nThreadGroupsTotal = nBlocks * nThreadGroupsPerBlock;
  unsigned int threadGroupIdx = blockDim.y * blockIdx.y + threadIdx.y;

  // Shared memory:
  // s_collisions: nThreadGroupsTotal bools
  //   size in bytes: (nThreadGroupsTotal + 3) / 4 * 4
  // s_neighborBuffer: IdxType[coreThreshold]
  //   size in bytes: coreThreshold * 4       TODO: Why not (coreThreshold - 1) * 4?
  // -> simultaneously used for s_interWarpUnionize: IdxType[blockDim.x / 32]
  //   size in bytes: (blockDim.x + 31) / 32 * 4
  // => size in bytes: max(coreThreshold, (blockDim.x + 31) / 32) * 4
  // s_neighborCount: IdxType
  //   size in bytes: 4
  // -> Total size per thread group in bytes:
  //   [(nThreadGroupsTotal + 3) / 4 + max(coreThreshold, (blockDim.x + 31) / 32) + 1] * 4 bytes
  extern __shared__ char sMem [];
  unsigned int sMemBytesPerThreadGroup = 4 * (
    (nThreadGroupsTotal + 3) / 4 +
    dhi_max(coreThreshold, (blockDim.x + 31) / 32) +
    1
  );

  static_assert(
    sizeof(IdxType)  == 4 &&
    alignof(IdxType) == 4 &&
    sizeof(bool) == 1, ""
  );
  bool * s_collisions        = (bool *)    (sMem                     + sMemBytesPerThreadGroup * threadIdx.y);
  IdxType * s_neighborBuffer = (IdxType *) ((char *)s_collisions     + (nThreadGroupsTotal + 3) / 4 * 4);
  volatile IdxType * s_interWarpUnionize = s_neighborBuffer;
  IdxType * s_neighborCount  = (IdxType *) ((char *)s_neighborBuffer + dhi_max(coreThreshold, (blockDim.x + 31) / 32) * 4);

  // clear all shared memory
  sharedMemZero(sMem, sMemBytesPerThreadGroup * blockDim.y);

  if (threadGroupIdx < n - beginStep) {
    // all threads in a thread group always examine the same point, but the
    //   cluster ids may diverge during the process
    IdxType ourPointIdx = beginStep + threadGroupIdx;
    IdxType myClusterId = ourPointIdx + clusters[ourPointIdx];

    __syncthreads();

    myClusterId = processPoints(
      coreMarkers, clusters, s_collisions,
      s_neighborBuffer, s_neighborCount,
      xs, ys, n,
      coreThreshold, rsq,
      beginStep, nThreadGroupsTotal < n - beginStep ? beginStep + nThreadGroupsTotal : n,
      ourPointIdx, myClusterId
    );

    __syncthreads();

    IdxType neighborCount = *s_neighborCount;

    if (neighborCount >= coreThreshold) {
      myClusterId = unionizeWithinThreadGroup(s_interWarpUnionize, clusters, myClusterId);

      __syncthreads();
  
      if (threadIdx.x == 0) {
        clusters[ourPointIdx] = myClusterId - ourPointIdx;
        coreMarkers[ourPointIdx] = true;
      }
    } else {
      for (int i = threadIdx.x; i < neighborCount; i += stride) {
        IdxType neighbor = s_neighborBuffer[i];
        if (neighbor < beginStep && coreMarkers[neighbor]) { clusters[ourPointIdx] = neighbor - ourPointIdx; break; }
      }
    }
  }

  __syncthreads();

  writeCollisions(collisionMatrix, s_collisions, nThreadGroupsTotal);
}

void allocateDeviceMemory(
  bool ** d_coreMarkers, IdxType ** d_clusters,
  unsigned int ** d_collisionMatrix,
  int nThreadGroups,
  IdxType n
) {
  CUDA_CHECK(hipMalloc(d_coreMarkers, n * sizeof(bool)))
  // TODO: Change later
  CUDA_CHECK(hipMemset(*d_coreMarkers, 0, n * sizeof(bool)))
  CUDA_CHECK(hipMalloc(d_clusters, n * sizeof(IdxType)))
  // TODO: Change later
  CUDA_CHECK(hipMemset(*d_clusters, 0, n * sizeof(IdxType)))

  unsigned int collisionPitch = (nThreadGroups + 31) / 32;
  CUDA_CHECK(hipMalloc(d_collisionMatrix, collisionPitch * nThreadGroups * sizeof(unsigned int)))
}


void unionizeCpu(std::vector<IdxType> & clusters) {
  std::vector<IdxType> stack;
  for (IdxType i = 0; i < clusters.size(); ++i) {
    IdxType child = i;
    IdxType parentOffset = clusters[child];
    if (parentOffset == 0) continue; // noise
    do {
      stack.push_back(child);
      child = child + parentOffset;
      parentOffset = clusters[child];
    } while (parentOffset);
    IdxType top = child;
    while (stack.size() > 0) {
      IdxType current = stack.back(); stack.pop_back();
      clusters[current] = top - current;
    }
  }
}

static __global__ void kernel_unionize(IdxType * clusters, IdxType n) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  IdxType strideBegin = 0;

  auto doWork = [&] {
    IdxType idx = strideBegin + tid;
    IdxType cl = idx;
    IdxType pOffset = clusters[idx];
    if (pOffset) {
      for (;;) {
        cl += pOffset;
        pOffset = clusters[cl];
        if (!pOffset) break;
      }
      IdxType top = cl;
      cl = idx;
      for (;;) {
        pOffset = clusters[cl];
        if (!pOffset) break;
        clusters[cl] = top - cl;
        cl += pOffset;
      }
    }
  };

  if (n > stride) for (; strideBegin < n - stride; strideBegin += stride) doWork();
  if (tid < n - strideBegin) doWork();
}

void findClusters(
  bool ** d_coreMarkers, IdxType ** d_clusters,
  float * xs, float * ys, IdxType n,
  IdxType coreThreshold, float rsq
) {
  constexpr int nBlocks = 6;
  constexpr int nThreadGroupsPerBlock = 32;
  constexpr int nThreadsPerBlock = 1024;

  int nThreadGroupsTotal = nBlocks * nThreadGroupsPerBlock;

  unsigned int sharedBytesPerBlock = nThreadGroupsPerBlock * 4 * (
    (nThreadGroupsTotal + 3) / 4 +
    dhi_max(coreThreshold, (nThreadGroupsPerBlock + 31) / 32) +
    1
  );

  unsigned int * d_collisionMatrix;
  allocateDeviceMemory(d_coreMarkers, d_clusters, &d_collisionMatrix, nThreadGroupsTotal, n);

  IdxType startPos = 0;
  for (;;) {
    kernel_clusterExpansion <<<dim3(1, nBlocks), dim3(nThreadsPerBlock / nThreadGroupsPerBlock, nThreadGroupsPerBlock), sharedBytesPerBlock >>> (
      *d_clusters, *d_coreMarkers, xs, ys, n, startPos, d_collisionMatrix, coreThreshold, rsq
    );
    unsigned int nCHThreads = nThreadGroupsTotal * nThreadGroupsTotal;
    unsigned int nCHThreadsPerBlock = 128;
    unsigned int nCHBlocks = (nCHThreads + nCHThreadsPerBlock - 1) / nCHThreadsPerBlock;
    kernel_handleCollisions <<<nCHBlocks, nCHThreadsPerBlock>>> (
      d_collisionMatrix, *d_clusters, *d_coreMarkers, n, startPos, nThreadGroupsTotal
    );

    //CUDA_CHECK(hipDeviceSynchronize())

    if (n - startPos <= nThreadGroupsTotal) break;
    startPos += nThreadGroupsTotal;
  }
  kernel_unionize <<<nBlocks, nThreadsPerBlock>>> (*d_clusters, n);
  CUDA_CHECK(hipGetLastError())
}

void unionizeGpu(IdxType * d_clusters, IdxType n) {
  constexpr unsigned int nBlocks = 6;
  constexpr unsigned int nThreadsPerBlock = 1024;

  kernel_unionize <<<dim3(nBlocks), dim3(nThreadsPerBlock)>>> (d_clusters, n);
  CUDA_CHECK(hipGetLastError())
  CUDA_CHECK(hipDeviceSynchronize())
}