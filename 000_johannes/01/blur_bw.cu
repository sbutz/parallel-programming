#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "get_image.h"
#include "write_image.h"

__global__ void bw_blur_kernel(
  float * input, float *output, int width, int height, int margin
) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= height || col >= width) return;

  int const lowerRow = row > margin ? row - margin : 0;
  int const upperRow = height - row > margin ? row + margin : height - 1;
  int const lowerCol = col > margin ? col - margin : 0;
  int const upperCol = width - col > margin ? col + margin : width - 1;
  float sum = 0.0;
  for (int i = lowerRow; i <= upperRow; ++i) {
    for (int j = lowerCol; j <= upperCol; ++j) {
      sum += input[i * width + j];
    }
  }
  float n = (float) (upperRow - lowerRow + 1) * (float) (upperCol - lowerCol + 1);
  output [row * width + col] = sum / n;
}


int main(int argc, char ** argv){
  int w=0, h=0, channel=0;
  float * data = NULL;
  float * in_d = NULL;
  float * out_d = NULL;
  float * out_h = NULL;
  const char * file_in = "minion_bw.jpg";
  const char * file_out = "minion_bw_blurred2.jpg";

  int const margin = 10;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  get_image(file_in, &data, &w , &h, &channel);

  int const in_size = sizeof(float) * w * h;
  int const out_size = sizeof(float) * w * h;

  out_h = (float *) malloc(out_size);

  hipEventRecord(start);
	hipMalloc((void **) &in_d, in_size);
	hipMemcpy(in_d, data, in_size, hipMemcpyHostToDevice);
	hipMalloc((void **) &out_d, out_size);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  dim3 dimGrid((w - 1) / 16 + 1, (h - 1) / 16 + 1, 1);
  dim3 dimBlock(16, 16, 1);
  hipEventRecord(start);
  bw_blur_kernel<<<dimGrid, dimBlock>>> (in_d, out_d, w, h, margin);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventRecord(start);
	hipMemcpy(out_h, out_d, out_size, hipMemcpyDeviceToHost);
  hipFree(in_d);
  hipFree(out_d);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  write_JPEG_file_bw (file_out, out_h, w, h);

  return 0;
}
