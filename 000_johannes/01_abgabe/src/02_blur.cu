#include "hip/hip_runtime.h"
#include "jpeg.h"
#include "util.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>

static constexpr std::size_t N_ITERATIONS = 100;

__global__ void Blur(unsigned char *inputImage, unsigned char *outputImage, int width, int height,
                     int channels, int margin) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = threadIdx.z;

    if (x < width && y < height) {
        int startX = max(x - margin, 0);
        int endX = min(x + margin, width);
        int startY = max(y - margin, 0);
        int endY = min(y + margin, height);

        float v = 0;
        for (int i = startY; i < endY; i++) {
            for (int j = startX; j < endX; j++) {
                v += inputImage[(i * width + j) * channels + channel];
            }
        }
        float n = (endX - startX) * (endY - startY);
        outputImage[(y * width + x) * channels + channel] = v / n;
    }
}

void Filter(const char *inputFilename, const char *outputFilename, std::size_t margin) {
    auto hInputImage = Jpeg::FromFile(inputFilename);
    auto height = hInputImage.GetHeight();
    auto width = hInputImage.GetWidth();
    auto channels = hInputImage.GetChannels();

    unsigned char *dInputImage, *dOutputImage;
    CUDA_ASSERT(hipMalloc((void **)&dInputImage, width * height * channels * sizeof(float)));
    CUDA_ASSERT(hipMalloc((void **)&dOutputImage, width * height * channels * sizeof(float)));
    CUDA_ASSERT(hipMemcpy(dInputImage, hInputImage.GetRawData(), width * height * channels,
                           hipMemcpyHostToDevice));

    dim3 blockSize(16, 16, channels);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    Blur<<<gridSize, blockSize>>>(dInputImage, dOutputImage, width, height, channels, margin);
    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    Jpeg hOutputImage{width, height, channels};
    CUDA_ASSERT(hipMemcpy(hOutputImage.GetRawData(), dOutputImage, width * height * channels,
                           hipMemcpyDeviceToHost));
    hOutputImage.Save(outputFilename);

    CUDA_ASSERT(hipFree(dInputImage));
    CUDA_ASSERT(hipFree(dOutputImage));
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " margin input.jpg output.jpg" << std::endl;
        return 1;
    }

    cudaInit();

    int margin = std::atoi(argv[1]);
    const char *inputFilename = argv[2];
    const char *outputFilename = argv[3];

    std::size_t input_size = std::atoi(argv[1]);
    for (auto i = 0; i < N_ITERATIONS; i++) {
        Filter(inputFilename, outputFilename, margin);
    }

    return 0;
}
