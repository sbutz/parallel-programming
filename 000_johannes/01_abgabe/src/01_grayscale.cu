#include "hip/hip_runtime.h"
#include "jpeg.h"
#include "util.h"
#include <hip/hip_runtime.h>
#include <iostream>

static constexpr std::size_t N_ITERATIONS = 100;

__global__ void RgbToGrayscale(unsigned char *inputImage, unsigned char *outputImage, int width,
                               int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        int rgb_idx = idx * 3;

        unsigned char r = inputImage[rgb_idx];
        unsigned char g = inputImage[rgb_idx + 1];
        unsigned char b = inputImage[rgb_idx + 2];

        unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);

        outputImage[idx] = gray;
    }
}

void Filter(const char *inputFilename, const char *outputFilename) {
    auto hInputImage = Jpeg::FromFile(inputFilename);
    auto height = hInputImage.GetHeight();
    auto width = hInputImage.GetWidth();
    auto channels = hInputImage.GetChannels();
    ASSERT(channels == 3, "Expecting an rgb image");

    unsigned char *dInputImage, *dOutputImage;
    CUDA_ASSERT(hipMalloc((void **)&dInputImage, width * height * channels));
    CUDA_ASSERT(hipMalloc((void **)&dOutputImage, width * height));

    CUDA_ASSERT(hipMemcpy(dInputImage, hInputImage.GetRawData(), width * height * channels,
                           hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    RgbToGrayscale<<<gridSize, blockSize>>>(dInputImage, dOutputImage, width, height);
    CUDA_ASSERT(hipGetLastError());
    CUDA_ASSERT(hipDeviceSynchronize());

    Jpeg hOutputImage{width, height, 1};
    CUDA_ASSERT(hipMemcpy(hOutputImage.GetRawData(), dOutputImage, width * height,
                           hipMemcpyDeviceToHost));
    hOutputImage.Save(outputFilename);

    CUDA_ASSERT(hipFree(dInputImage));
    CUDA_ASSERT(hipFree(dOutputImage));
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " input.jpg output.jpg" << std::endl;
        return 1;
    }

    cudaInit();

    const char *inputFilename = argv[1];
    const char *outputFilename = argv[2];

    std::size_t input_size = std::atoi(argv[1]);
    for (auto i = 0; i < N_ITERATIONS; i++) {
        Filter(inputFilename, outputFilename);
    }

    return 0;
}
