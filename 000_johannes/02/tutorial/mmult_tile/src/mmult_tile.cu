#include <hip/hip_runtime.h>

#include <stdio.h>

using std::size_t;

constexpr size_t M = 1000;
constexpr size_t K = 2000;
constexpr size_t N = 1500;
constexpr size_t blocksize = 32;

__global__ void mmult_tile_kernel(
	float * C, float * A, float * B, size_t m, size_t k, size_t n
) {
	size_t r = blockIdx.x;
  size_t c = blockIdx.y;
  size_t threadR = threadIdx.x / blocksize;
  size_t threadC = threadIdx.x % blocksize;

  __shared__ float As[blocksize * blocksize], Bs[blocksize * blocksize];

  size_t baseIdxA = r * blocksize * k;
  size_t baseIdxB = c * blocksize;

  float s = 0.0f;
  for (size_t blkIdx = 0; blkIdx < k; blkIdx += blocksize) {
    As[threadR * blocksize + threadC] =
      r * blocksize + threadR < m &&
      blkIdx + threadC < k ?
        A[baseIdxA + threadR * k + threadC] 
      :
        0.0f;
    Bs[threadR * blocksize + threadC] = 
      blkIdx + threadR < k &&
      c * blocksize + threadC < n ?
        B[baseIdxB + threadR * n + threadC]
      :
        0.0f;
    __syncthreads();

    baseIdxA += blocksize;
    baseIdxB += blocksize * n;

    for (size_t dotIdx = 0; dotIdx < blocksize; ++dotIdx) {
      s += As[threadR * blocksize + dotIdx] * Bs[dotIdx * blocksize + threadC]; 
    }
    __syncthreads();
  }

  if (
    r * blocksize + threadR < m && c * blocksize + threadC < n
  ) {
    C[n * (r * blocksize + threadR) + (c * blocksize + threadC)] = s;
  }
}

int main () {
  float * A = (float *) malloc (M * K * sizeof(float));
  float * B = (float *) malloc (K * N * sizeof(float));
  float * C = (float *) malloc (M * N * sizeof(float));

  for (size_t i = 0; i < M * K; ++i) A[i] = (float) i;
  for (size_t i = 0; i < K * N; ++i) B[i] = (float) i;

  float * deviceA, * deviceB, * deviceC;
	hipMalloc((void **) &deviceA, M * K * sizeof(float));
	hipMalloc((void **) &deviceB, K * N * sizeof(float));
  hipMalloc((void **) &deviceC, M * N * sizeof(float));

	// kopiere die Daten auf die GPU
	hipMemcpy(deviceA, A, M * K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, B, K * N * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  dim3 dimGrid((M - 1)/32 + 1, (N - 1) / 32 + 1, 1);
  dim3 dimBlock(32 * 32);
  hipEventRecord(start);
  mmult_tile_kernel<<<dimGrid, dimBlock>>> (deviceC, deviceA, deviceB, M, K, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipMemcpy(C, deviceC, M * N * sizeof(float), hipMemcpyDeviceToHost);

	float calcTime = 0;
	hipEventElapsedTime(&calcTime, start, stop);
	printf("Calc time %.4f ms\n", calcTime);

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  // check
  for (size_t i = 0; i < M; ++i)
    for (size_t j = 0; j < N; ++j) {
      float s = 0.0f;
      for (size_t k = 0; k < K; ++k) {
        s += ((float)i * K + k) * ((float) j + (float)k * N);
      }
      if (abs(C[N * i + j]/s - 1) > 0.0001) {
        printf("Wrong result at %lu %lu -- expected: %f, actual: %f", i, j, s, C[N*i +j]); exit(0);    
      }
    }
  return 0;
}