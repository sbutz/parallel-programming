#include <hip/hip_runtime.h>

#include <stdio.h>

using std::size_t;

constexpr size_t M = 1000;
constexpr size_t K = 2000;
constexpr size_t N = 1500;

template <size_t BM, size_t BN, size_t BK, size_t TM>
__global__ void mmult_1dblocktile_kernel(
	float * C, float * A, float * B, size_t m, size_t k, size_t n
) {
	size_t r = blockIdx.y;
  size_t c = blockIdx.x;
  size_t threadR = threadIdx.x / BN;
  size_t threadC = threadIdx.x % BN;

  __shared__ float As[BM * BK], Bs[BK * BN];

  size_t baseIdxA = r * BM * k;
  size_t baseIdxB = c * BN;

  size_t innerRowA = threadIdx.x / BK;
  size_t innerColA = threadIdx.x % BK;
  size_t innerRowB = threadIdx.x / BN;
  size_t innerColB = threadIdx.x % BN;

  float threadResults[TM] = { 0.0f };
  for (size_t blkIdx = 0; blkIdx < k; blkIdx += BK) {
    As[innerRowA * BK + innerColA] =
      r * BM + innerRowA < m &&
      blkIdx + innerColA < k ?
        A[baseIdxA + innerRowA * k + innerColA] 
      :
        0.0f;
    Bs[innerRowB * BN + innerColB] = 
      blkIdx + innerRowB < k &&
      c * BN + innerColB < n ?
        B[baseIdxB + innerRowB * n + innerColB]
      :
        0.0f;
    __syncthreads();

    baseIdxA += BK;
    baseIdxB += BK * n;

    for (size_t dotIdx = 0; dotIdx < BK; ++dotIdx) {
      float tmpB = Bs[dotIdx * BN + threadC];
      for (size_t resIdx = 0; resIdx < TM; ++resIdx) {
        threadResults[resIdx] += As[(threadR * TM + resIdx) * BK + dotIdx] * tmpB;
      } 
    }
    __syncthreads();
  }

  for (size_t resIdx = 0; resIdx < TM; ++resIdx) {
    if (
      r * BM + threadR * TM + resIdx < m &&
      c * BN + threadC < n
    ) {
      C [(r * BM + threadR * TM + resIdx) * n + c * BN + threadC] = threadResults[resIdx];
    }
  }
}

int main () {
  float * A = (float *) malloc (M * K * sizeof(float));
  float * B = (float *) malloc (K * N * sizeof(float));
  float * C = (float *) malloc (M * N * sizeof(float));

  for (size_t i = 0; i < M * K; ++i) A[i] = (float) i;
  for (size_t i = 0; i < K * N; ++i) B[i] = (float) 1;

  float * deviceA, * deviceB, * deviceC;
	hipMalloc((void **) &deviceA, M * K * sizeof(float));
	hipMalloc((void **) &deviceB, K * N * sizeof(float));
  hipMalloc((void **) &deviceC, M * N * sizeof(float));

	// kopiere die Daten auf die GPU
	hipMemcpy(deviceA, A, M * K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, B, K * N * sizeof(float), hipMemcpyHostToDevice);
//	cudaMemcpy(deviceC, C, M * N * sizeof(float), cudaMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  constexpr size_t BM = 64;
  constexpr size_t BN = 64;
  constexpr size_t BK = 8;
  constexpr size_t TM = 8;

  dim3 dimGrid((N - 1) / BN + 1, (M - 1) / BM + 1, 1);
  dim3 dimBlock(BM * BN / TM);
  hipEventRecord(start);
  mmult_1dblocktile_kernel<BM, BN, BK, TM> <<<dimGrid, dimBlock>>> (deviceC, deviceA, deviceB, M, K, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipMemcpy(C, deviceC, M * N * sizeof(float), hipMemcpyDeviceToHost);

	float calcTime = 0;
	hipEventElapsedTime(&calcTime, start, stop);
	printf("Calc time %.4f ms\n", calcTime);

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  // check
  for (size_t i = 0; i < M; ++i)
    for (size_t j = 0; j < N; ++j) {
      float s = 0.0f;
      for (size_t k = 0; k < K; ++k) {
        s += A[i * K + k] * B[j + k * N];
      }
      if (abs(C[N * i + j]/s - 1) > 0.0001) {
        printf("Wrong result at %lu %lu -- expected: %f, actual: %f", i, j, s, C[N*i +j]); exit(0);    
      }
    }
  return 0;
}