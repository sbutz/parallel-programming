#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__global__ void VecAddKernel(const float *a, const float *b,  float *c, const int n) {
	int i = threadIdx.x+blockDim.x*blockIdx.x;
	if(i<n) c[i] = a[i] + b[i];
}

void vecAdd(float* h_A, float* h_B, float* h_C, int n)
{

	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	hipMalloc((void **) &d_A, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_B, size);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_C, size);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float memory_time1 = 0;
	hipEventElapsedTime(&memory_time1, start, stop);

	dim3 DimGrid((n-1)/256 + 1, 1, 1);
	dim3 DimBlock(256, 1, 1);

	hipEventRecord(start);
	VecAddKernel<<<DimGrid,DimBlock>>>(d_A, d_B, d_C, n);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float kernel_time = 0;
	hipEventElapsedTime(&kernel_time, start, stop);


	hipEventRecord(start);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float memory_time2 = 0;
	hipEventElapsedTime(&memory_time2, start, stop);
	printf("memory time 1 %.2f,  memory time 1 %.2f, kernel time %.2f\n", memory_time1, memory_time2, kernel_time);

}

int main() {
	int n = 1<<20;

	float *h_A, *h_B, *h_C;
	h_A = (float*)malloc(sizeof(float)*n);
	h_B = (float*)malloc(sizeof(float)*n);

	h_C = (float*)malloc(sizeof(float)*n);

	for(int i = 0; i<n; i++) {
		h_A[i] = (float)(rand()%100);
		h_B[i]= (float)(rand()%200);
	}

	vecAdd(h_A, h_B, h_C,n);

	free(h_A);
	free(h_B);
	free(h_C);

}
